#include "hip/hip_runtime.h"
#include "BVH.cuh"

#include "Utility\CUDA\CUDAHelper.cuh"
#include "Utility/Includes/GLMIncludes.h"

// Returns the highest differing bit of i and i+1
__device__ uint HighestBit(uint i, uint64* morton)
{
	return morton[i] ^ morton[i + 1];
}

__global__ void BuildTree(uint n, uint innerSize, BVHData* data, Node* nodes, uint64* mortonCodes)
{
	const uint index = getGlobalIdx_1D_1D();
	if (index >= n) {
		return;
	}

	uint nodeOffset = innerSize + index;
	Node* nodePointer = nodes + nodeOffset;
	Node currentNode = nodes[nodeOffset];
	
	while (true) {
		// Allow only one thread to process a node
		if (atomicAdd(&nodePointer->atomic, 1) != 1)
			return;

		// Set bounding box if the node is not a leaf
		if (nodeOffset < innerSize)
		{
			const BoundingBox boxLeft = nodes[currentNode.childLeft].box;
			const BoundingBox boxRight = nodes[currentNode.childRight].box;

			currentNode.box.max = glm::max(boxLeft.max, boxRight.max);
			currentNode.box.min = glm::min(boxLeft.min, boxRight.min);

			nodes[nodeOffset] = currentNode;
		}

		if (currentNode.rangeLeft == 0 && currentNode.rangeRight == innerSize) {
			data->root = nodeOffset;
			return;
		}

		Node* parentPointer;

		if (currentNode.rangeLeft == 0 || currentNode.rangeRight < innerSize && 
			HighestBit(currentNode.rangeLeft - 1, mortonCodes) > HighestBit(currentNode.rangeRight, mortonCodes))
		{

			// parent = right, set parent left child and range to node		
			parentPointer = nodes + currentNode.rangeRight;
			Node parent = *parentPointer;
			parent.childLeft = nodeOffset;
			parent.rangeLeft = currentNode.rangeLeft;
			*parentPointer = parent;

		}
		else
		{

			// parent = left -1, set parent right child and range to node
			parentPointer = nodes + (currentNode.rangeLeft - 1);
			Node parent = *parentPointer;
			parent.childRight = nodeOffset;
			parent.rangeRight = currentNode.rangeRight;
			*parentPointer = parent;
			
		}

		nodePointer = parentPointer;
		nodeOffset =  nodePointer - nodes;
		currentNode = *nodePointer;
	}
}


__global__ void Reset(uint n, uint innerSize, Node* nodes, Face* faces, Vertex* vertices)
{
	const uint index = getGlobalIdx_1D_1D();

	if (index >= n) {
		return;
	}


	const uint leafOffset = innerSize + index;

	//set the inner node
	if (index < innerSize) {
		Node temp;
		temp.atomic = 0; //inner nodes are not visited
		temp.childLeft = leafOffset;
		temp.childRight = leafOffset + 1;
		nodes[index] = temp;
	}


	const glm::uvec3 ind = faces[index].indices;

	// Expand bounds using min/max functions
	const glm::vec3 pos0 = vertices[ind.x].position;
	const glm::vec3 pos1 = vertices[ind.y].position;
	const glm::vec3 pos2 = vertices[ind.z].position;

	glm::vec3 max = pos0;
	glm::vec3 min = pos0;

	max = glm::max(pos1, max);
	min = glm::min(pos1, min);

	max = glm::max(pos2, max);
	min = glm::min(pos2, min);

	//set the leaf node
	Node temp;
	temp.rangeLeft = index;
	temp.rangeRight = index;
	temp.atomic = 1; // To allow the next thread to process
	temp.faceID = index; //set triangle
	temp.box.max = max;
	temp.box.min = min;
	temp.childLeft = static_cast<uint>(-1); //set termination
	temp.childRight = static_cast<uint>(-1); //set termination

	nodes[leafOffset] = temp;
}