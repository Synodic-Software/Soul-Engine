#include "hip/hip_runtime.h"
#include "Object.cuh"
#include "Utility\CUDA\CUDAHelper.cuh"

#define TINYOBJLOADER_IMPLEMENTATION
#include <tiny_obj_loader.h>

#include <unordered_map>

Object::Object(){

	verticeAmount = 0;
	faceAmount = 0;
	materialSize = 0;
	localSceneIndex = 0;
	ready = false;

	xyzPosition = glm::vec3(0);

	vertices = NULL;
	faces = NULL;
	materialP = NULL;
}
Object::Object(glm::vec3 pos, std::string name, Material* mat){

	verticeAmount = 0;
	faceAmount = 0;
	materialSize = 1;
	localSceneIndex = 0;
	ready = false;

	xyzPosition = glm::vec3(0);

	vertices = NULL;
	faces = NULL;
	CudaCheck(hipMallocManaged((void**)&materialP, materialSize*sizeof(Material*)));
	materialP[0] = mat;

	xyzPosition = pos;
	ExtractFromFile(name.c_str());
}

void Object::AddVertices(Vertex* vertices, uint vSize){

}
void Object::AddFaces(Face* vertices, uint fSize){

}
void Object::ExtractFromFile(const char* name){



	tinyobj::attrib_t attrib;
	std::vector<tinyobj::shape_t> shapes;
	std::vector<tinyobj::material_t> materials;
	std::string err;

	if (!tinyobj::LoadObj(&attrib, &shapes, &materials, &err, name)) {
		throw std::runtime_error(err);
	}

	assert(shapes.size() == 1);

	verticeAmount = attrib.vertices.size() / 3;
	faceAmount = shapes[0].mesh.indices.size() / 3;

	glm::vec3 max = glm::vec3(attrib.vertices[0], attrib.vertices[1], attrib.vertices[2]);
	glm::vec3 min = max;

	CudaCheck(hipDeviceSynchronize());

	CudaCheck(hipMallocManaged((void**)&vertices,
		verticeAmount*sizeof(Vertex)));

	CudaCheck(hipMallocManaged((void**)&faces,
		faceAmount*sizeof(Face)));

	CudaCheck(hipDeviceSynchronize());



	std::unordered_map<Vertex, int> uniqueVertices = {};

	const auto& shape = shapes[0];

	for (size_t f = 0; f < shape.mesh.indices.size() / 3; f++) {


		//grab commenly used variables
		tinyobj::index_t id0 = shape.mesh.indices[3 * f + 0];
		tinyobj::index_t id1 = shape.mesh.indices[3 * f + 1];
		tinyobj::index_t id2 = shape.mesh.indices[3 * f + 2];

		int current_material_id = shape.mesh.material_ids[f];

		faces[f].indices.x = id0.vertex_index;
		vertices[id0.vertex_index].position.x = attrib.vertices[id0.vertex_index * 3 + 0];
		vertices[id0.vertex_index].position.y = attrib.vertices[id0.vertex_index * 3 + 1];
		vertices[id0.vertex_index].position.z = attrib.vertices[id0.vertex_index * 3 + 2];

		vertices[id0.vertex_index].textureCoord.x = attrib.texcoords[id0.texcoord_index * 2 + 0];
		vertices[id0.vertex_index].textureCoord.y = 1.0f - attrib.texcoords[id0.texcoord_index * 2 + 1];

		vertices[id0.vertex_index].normal.x = attrib.normals[id0.normal_index * 3 + 0];
		vertices[id0.vertex_index].normal.y = attrib.normals[id0.normal_index * 3 + 1];
		vertices[id0.vertex_index].normal.z = attrib.normals[id0.normal_index * 3 + 2];

		vertices[id0.vertex_index].position += xyzPosition;
		max = glm::max(vertices[id0.vertex_index].position, max);
		min = glm::min(vertices[id0.vertex_index].position, min);

		///////////////////

		faces[f].indices.y = id1.vertex_index;
		vertices[id1.vertex_index].position.x = attrib.vertices[id1.vertex_index * 3 + 0];
		vertices[id1.vertex_index].position.y = attrib.vertices[id1.vertex_index * 3 + 1];
		vertices[id1.vertex_index].position.z = attrib.vertices[id1.vertex_index * 3 + 2];

		vertices[id1.vertex_index].textureCoord.x = attrib.texcoords[id1.texcoord_index * 2 + 0];
		vertices[id1.vertex_index].textureCoord.y = 1.0f - attrib.texcoords[id1.texcoord_index * 2 + 1];

		vertices[id1.vertex_index].normal.x = attrib.normals[id1.normal_index * 3 + 0];
		vertices[id1.vertex_index].normal.y = attrib.normals[id1.normal_index * 3 + 1];
		vertices[id1.vertex_index].normal.z = attrib.normals[id1.normal_index * 3 + 2];

		vertices[id1.vertex_index].position += xyzPosition;
		max = glm::max(vertices[id1.vertex_index].position, max);
		min = glm::min(vertices[id1.vertex_index].position, min);

		///////////////////

		faces[f].indices.z = id2.vertex_index;
		vertices[id2.vertex_index].position.x = attrib.vertices[id2.vertex_index * 3 + 0];
		vertices[id2.vertex_index].position.y = attrib.vertices[id2.vertex_index * 3 + 1];
		vertices[id2.vertex_index].position.z = attrib.vertices[id2.vertex_index * 3 + 2];

		vertices[id2.vertex_index].textureCoord.x = attrib.texcoords[id2.texcoord_index * 2 + 0];
		vertices[id2.vertex_index].textureCoord.y = 1.0f - attrib.texcoords[id2.texcoord_index * 2 + 1];

		vertices[id2.vertex_index].normal.x = attrib.normals[id2.normal_index * 3 + 0];
		vertices[id2.vertex_index].normal.y = attrib.normals[id2.normal_index * 3 + 1];
		vertices[id2.vertex_index].normal.z = attrib.normals[id2.normal_index * 3 + 2];

		vertices[id2.vertex_index].position += xyzPosition;
		max = glm::max(vertices[id2.vertex_index].position, max);
		min = glm::min(vertices[id2.vertex_index].position, min);

		faces[f].materialPointer = materialP[0];
	}




	//LOG(TRACE, "\nINDICES: " << faceAmount << std::endl;
	//for (int i = 0; i < faceAmount; i++){
	//	LOG(TRACE,("%i ", faces[i].indices.x);
	//	LOG(TRACE,("%i ", faces[i].indices.y);
	//	LOG(TRACE,("%i \n", faces[i].indices.z);
	//}

	//LOG(TRACE, "\nVERTICES: " << verticeAmount << std::endl;

	//for (int i = 0; i < verticeAmount; i++){
	//	LOG(TRACE, "\n	Positions: "  << std::endl;

	//	LOG(TRACE,("%f ", vertices[i].position.x);
	//	LOG(TRACE,("%f ", vertices[i].position.y);
	//	LOG(TRACE,("%f \n", vertices[i].position.z);

	//	LOG(TRACE,"\n	Normals: " << std::endl;

	//	LOG(TRACE,("%f ", vertices[i].normal.x);
	//	LOG(TRACE,("%f ", vertices[i].normal.y);
	//	LOG(TRACE,("%f \n", vertices[i].normal.z);

	//	LOG(TRACE, "\n	TexCoords: " << std::endl;

	//	LOG(TRACE,("%f ", vertices[i].textureCoord.x);
	//	LOG(TRACE,("%f \n", vertices[i].textureCoord.y);
	//}

	box.max = max;
	box.min = min;

	int device=0;
	CudaCheck(hipGetDevice(&device));

	/*CudaCheck(hipMemAdvise(vertices, verticeAmount*sizeof(Vertex), hipMemAdviseSetAccessedBy, device));
	CudaCheck(hipMemPrefetchAsync(vertices, verticeAmount*sizeof(Vertex), device));

	CudaCheck(hipMemAdvise(faces, faceAmount*sizeof(Face), hipMemAdviseSetAccessedBy, device));
	CudaCheck(hipMemPrefetchAsync(faces, faceAmount*sizeof(Face), device));*/
}