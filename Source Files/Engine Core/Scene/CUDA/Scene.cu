#include "hip/hip_runtime.h"
#include "Scene.cuh"

#define RAY_BIAS_DISTANCE 0.0002f 
#define BVH_STACK_SIZE 64
#define DYNAMIC_FETCH_THRESHOLD 20          // If fewer than this active, fetch new rays

#include "Utility\CUDA\CUDAHelper.cuh"
#include "Utility\Logger.h"

#include "Algorithms\Morton Code\MortonCode.h"

#include <thrust/fill.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/remove.h>
#include <thrust/functional.h>

Scene::Scene()
{
	objectsSize = 0;
	allocatedObjects = 1;
	allocatedSize = 0;

	compiledSize = 0;
	newFaceAmount = 0;
	CudaCheck(hipMallocManaged((void**)&objectList,
		allocatedObjects*sizeof(Object*)));

	objectsToRemove.clear();

	//give the addresses for the data
	bvh = new BVH(&faceIds, &mortonCodes);

	sky = new Sky("Starmap.png");
}


Scene::~Scene()
{

	CudaCheck(hipFree(objectBitSetup)); // hold a true for the first indice of each object
	CudaCheck(hipFree(objIds)); //points to the object
	CudaCheck(hipFree(faceIds));
	CudaCheck(hipFree(mortonCodes));

	//Variables concerning object storage

	CudaCheck(hipFree(objectList));
	CudaCheck(hipFree(objectRemoval));

	delete bvh;
}


struct is_scheduled
{
	__host__ __device__
		bool operator()(const Object* x)
	{
		return (x->requestRemoval);
	}
};


__global__ void FillBool(const uint n, bool* jobs, bool* fjobs, Face** faces, uint* objIds, Object** objects){


	uint index = getGlobalIdx_1D_1D();


	if (index < n){
		if (objects[objIds[index] - 1]->requestRemoval){
			jobs[index] = true;
		}
		else{
			jobs[index] = false;
		}
		if (faces[index]->objectPointer->requestRemoval){
			fjobs[index] = true;
		}
		else{
			fjobs[index] = false;
		}
	}
}

__global__ void GetFace(const uint n, uint* objIds, Object** objects, Face** faces, const uint offset){


	uint index = getGlobalIdx_1D_1D();


	if (index >= n){
		return;
	}

	Object* obj = objects[objIds[offset + index] - 1];
	faces[offset + index] = obj->faces + (index - obj->localSceneIndex);
	faces[offset + index]->objectPointer = obj;

}


//add all new objects into the scene's arrays
__host__ bool Scene::Compile(){

	uint amountToRemove = objectsToRemove.size();

	if (newFaceAmount > 0 || amountToRemove > 0){

		//bitSetup only has the first element of each object flagged
		//this extends that length and copies the previous results as well

		uint newSize = compiledSize + newFaceAmount;
		uint indicesToRemove = 0;
		uint removedOffset = 0;
		if (amountToRemove > 0){

			bool* markers;
			bool* faceMarkers;
			CudaCheck(hipMalloc((void**)&markers, compiledSize * sizeof(bool)));
			CudaCheck(hipMalloc((void**)&faceMarkers, compiledSize * sizeof(bool)));

			thrust::device_ptr<bool> tempPtr = thrust::device_pointer_cast(markers);
			thrust::device_ptr<bool> faceTempPtr = thrust::device_pointer_cast(faceMarkers);


			//variables from the scene to the kernal

			uint blockSize = 64;
			uint gridSize = (compiledSize + blockSize - 1) / blockSize;

			//fill the mask with 1s or 0s
			FillBool << <gridSize, blockSize >> >(compiledSize, markers, faceMarkers, faceIds, objIds, objectList);
			CudaCheck(hipPeekAtLastError());
			CudaCheck(hipDeviceSynchronize());


			//remove the requested
			thrust::device_ptr<bool> bitPtr = thrust::device_pointer_cast(objectBitSetup);

			thrust::device_ptr<bool> newEnd = thrust::remove_if(bitPtr, bitPtr + compiledSize, tempPtr, thrust::identity<bool>());
			CudaCheck(hipDeviceSynchronize());

			indicesToRemove = bitPtr + compiledSize - newEnd;
			newSize = newSize - indicesToRemove;
			//objpointers
			thrust::device_ptr<uint> objPtr = thrust::device_pointer_cast(objIds);

			thrust::remove_if(objPtr, objPtr + compiledSize, tempPtr, thrust::identity<bool>());
			CudaCheck(hipDeviceSynchronize());

			//faces
			thrust::device_ptr<Face*> facePtr = thrust::device_pointer_cast(faceIds);

			thrust::remove_if(facePtr, facePtr + compiledSize, faceTempPtr, thrust::identity<bool>());
			CudaCheck(hipDeviceSynchronize());

			//actual object list
			thrust::device_ptr<Object*> objectsPtr = thrust::device_pointer_cast(objectList);

			thrust::remove_if(objectsPtr, objectsPtr + objectsSize, is_scheduled());

			CudaCheck(hipDeviceSynchronize());
			CudaCheck(hipFree(markers));
			CudaCheck(hipFree(faceMarkers));
		}

		if (newFaceAmount > 0){

			if (allocatedSize < newSize){
				Face** faceTemp;
				uint* objTemp;
				bool* objectBitSetupTemp;

				allocatedSize = glm::max(uint(allocatedSize * 1.5f), newSize);

				CudaCheck(hipMalloc((void**)&faceTemp, allocatedSize * sizeof(Face*)));
				CudaCheck(hipMalloc((void**)&objTemp, allocatedSize * sizeof(uint)));
				CudaCheck(hipMalloc((void**)&objectBitSetupTemp, allocatedSize * sizeof(bool)));
				CudaCheck(hipFree(mortonCodes));
				CudaCheck(hipMalloc((void**)&mortonCodes, allocatedSize * sizeof(uint64)));

				CudaCheck(hipMemcpy(faceTemp, faceIds, compiledSize*sizeof(Face*), hipMemcpyDeviceToDevice));
				CudaCheck(hipFree(faceIds));
				faceIds = faceTemp;

				CudaCheck(hipMemcpy(objTemp, objIds, compiledSize*sizeof(uint), hipMemcpyDeviceToDevice));
				CudaCheck(hipFree(objIds));
				objIds = objTemp;

				CudaCheck(hipMemcpy(objectBitSetupTemp, objectBitSetup, compiledSize*sizeof(bool), hipMemcpyDeviceToDevice));
				CudaCheck(hipFree(objectBitSetup));
				objectBitSetup = objectBitSetupTemp;

			}

			CudaCheck(hipDeviceSynchronize());
			removedOffset = compiledSize - indicesToRemove;
			//for each new object, (all at the end of the array) fill with falses.
			thrust::device_ptr<bool> bitPtr = thrust::device_pointer_cast(objectBitSetup);
			thrust::fill(bitPtr + removedOffset, bitPtr + newSize, (bool)false);

			CudaCheck(hipDeviceSynchronize());

		}


		CudaCheck(hipDeviceSynchronize());

		//flag the first and setup state of life (only time iteration through objects should be done)
		uint l = 0;
		for (uint i = 0; i < objectsSize; i++){
			if (!objectList[i]->ready){
				CudaCheck(hipMemset(objectBitSetup + l, true, sizeof(bool)));
				objectList[i]->ready = true;
			}
			objectList[i]->localSceneIndex = l;
			l += objectList[i]->faceAmount;
		}

		if (newFaceAmount > 0){

			thrust::device_ptr<bool> bitPtr = thrust::device_pointer_cast(objectBitSetup);
			thrust::device_ptr<uint> objPtr = thrust::device_pointer_cast(objIds);
			CudaCheck(hipDeviceSynchronize());

			thrust::inclusive_scan(bitPtr, bitPtr + newSize, objPtr);
			CudaCheck(hipDeviceSynchronize());


			uint blockSize = 64;
			uint gridSize = ((newSize - removedOffset) + blockSize - 1) / blockSize;

			GetFace << <gridSize, blockSize >> >(newSize - removedOffset, objIds, objectList, faceIds, removedOffset);
			CudaCheck(hipPeekAtLastError());
			CudaCheck(hipDeviceSynchronize());

		}
		CudaCheck(hipDeviceSynchronize());

		//change the indice count of the scene
		compiledSize = newSize;
		newFaceAmount = 0;
		objectsToRemove.clear();
		return true;

	}
	else{
		return false;
	}


}


__host__ void Scene::Build(float deltaTime){

	int device;
	CudaCheck(hipGetDevice(&device));

	//CudaCheck(hipMemAdvise(objectList, objectsSize*sizeof(Object*)*sizeof(Vertex), hipMemAdviseSetAccessedBy, device));
	//CudaCheck(hipMemPrefetchAsync(objectList, objectsSize*sizeof(Object*)*sizeof(Vertex), device, 0));

	bool b = Compile();

	//calculate the morton code for each triangle

	uint blockSize = 64;
	uint gridSize = (compiledSize + blockSize - 1) / blockSize;

	CudaCheck(hipDeviceSynchronize());

	//CudaCheck(hipMemAdvise(objectList, objectsSize*sizeof(Object*)*sizeof(Vertex), hipMemAdviseSetAccessedBy, device));
	//CudaCheck(hipMemPrefetchAsync(objectList, objectsSize*sizeof(Object*)*sizeof(Vertex), device, 0));

	MortonCode::Compute << <gridSize, blockSize >> >(compiledSize, mortonCodes, faceIds, objectList, sceneBox);


	CudaCheck(hipPeekAtLastError());
	CudaCheck(hipDeviceSynchronize());
	thrust::device_ptr<uint64_t> keys(mortonCodes);
	thrust::device_ptr<Face*> values(faceIds);


	CudaCheck(hipDeviceSynchronize());

	hipEvent_t start, stop;
	float time;
	CudaCheck(hipEventCreate(&start));
	CudaCheck(hipEventCreate(&stop));
	CudaCheck(hipEventRecord(start, 0));

	CudaCheck(hipDeviceSynchronize());

	thrust::sort_by_key(keys, keys + compiledSize, values);           

	CudaCheck(hipDeviceSynchronize());

	CudaCheck(hipEventRecord(stop, 0));
	CudaCheck(hipEventSynchronize(stop));
	CudaCheck(hipEventElapsedTime(&time, start, stop));
	CudaCheck(hipEventDestroy(start));
	CudaCheck(hipEventDestroy(stop));

	S_LOG_TRACE("     Sorting Execution: " , time , "ms");

	bvh->Build(compiledSize);

}


__host__ uint Scene::AddObject(Object* obj){

	//if the size of objects stores increases, double the available size pool;
	if (objectsSize == allocatedObjects){

		Object** objectsTemp;
		allocatedObjects *= 2;

		if (allocatedObjects == 0){
			allocatedObjects = 1;
		}

		CudaCheck(hipMallocManaged((void**)&objectsTemp, allocatedObjects * sizeof(Object*)));

		CudaCheck(hipMemcpy(objectsTemp, objectList, objectsSize*sizeof(Object*), hipMemcpyDeviceToDevice));
		CudaCheck(hipFree(objectList));
		objectList = objectsTemp;
	}


	//update the scene's bounding volume

	sceneBox.max = glm::max(sceneBox.max, obj->box.max);
	sceneBox.min = glm::min(sceneBox.min, obj->box.min);

	//add the reference as the new object and increase the object count by 1
	CudaCheck(hipDeviceSynchronize());
	objectList[objectsSize] = obj;
	objectsSize++;
	newFaceAmount += obj->faceAmount;
	return 0;
}

__host__ bool Scene::RemoveObject(const uint& tag){
	objectRemoval[tag] = true;
	return true;
}
