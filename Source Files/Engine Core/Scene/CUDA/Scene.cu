#include "hip/hip_runtime.h"
#include "Scene.cuh"

#define RAY_BIAS_DISTANCE 0.0002f 
#define BVH_STACK_SIZE 64
#define DYNAMIC_FETCH_THRESHOLD 20          // If fewer than this active, fetch new rays

#include "Utility\CUDA\CUDAHelper.cuh"
#include "Utility\Logger.h"

#include "Algorithms\Morton Code\MortonCode.h"

#include <thrust/fill.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/remove.h>
#include <thrust/functional.h>

Scene::Scene()
{

	objects = nullptr;
	mortonCodes = nullptr;
	faces = nullptr;
	vertices = nullptr;
	materials = nullptr;
	tets = nullptr;

	faceAmount = 0;
	vertexAmount = 0;
	tetAmount = 0;
	materialAmount = 0;
	objectAmount = 0;

	faceAllocated = 0;
	vertexAllocated = 0;
	tetAllocated = 0;
	materialAllocated = 0;
	objectAllocated = 0;

	//
	bvhHost = new BVH();
	CudaCheck(hipMalloc((void **)&bvh, sizeof(BVH)));

	Sky* skyHost = new Sky("Starmap.png");
	CudaCheck(hipMalloc((void **)&sky, sizeof(Sky)));
	CudaCheck(hipMemcpy(sky, skyHost, sizeof(Sky), hipMemcpyHostToDevice));
}


Scene::~Scene()
{

	CudaCheck(hipFree(mortonCodes));
	CudaCheck(hipFree(faces));
	CudaCheck(hipFree(vertices));
	CudaCheck(hipFree(tets));
	CudaCheck(hipFree(materials));
	CudaCheck(hipFree(objects));

	delete bvhHost;

	CudaCheck(hipFree(bvh));
	CudaCheck(hipFree(sky));

}

__host__ void Scene::Build(float deltaTime) {

	Compile();

	//calculate the morton code for each triangle
	uint blockSize = 64;
	uint gridSize = (faceAmount + blockSize - 1) / blockSize;

	CudaCheck(hipDeviceSynchronize());

	MortonCode::Compute << <gridSize, blockSize >> > (faceAmount, mortonCodes, faces, vertices, sceneBox);

	CudaCheck(hipPeekAtLastError());
	CudaCheck(hipDeviceSynchronize());

	thrust::device_ptr<uint64_t> keys(mortonCodes);
	thrust::device_ptr<Face> values(faces);

	CudaCheck(hipDeviceSynchronize());

	hipEvent_t start, stop;
	float time;
	CudaCheck(hipEventCreate(&start));
	CudaCheck(hipEventCreate(&stop));
	CudaCheck(hipEventRecord(start, 0));

	CudaCheck(hipDeviceSynchronize());

	thrust::sort_by_key(keys, keys + faceAmount, values);

	CudaCheck(hipDeviceSynchronize());

	CudaCheck(hipEventRecord(stop, 0));
	CudaCheck(hipEventSynchronize(stop));
	CudaCheck(hipEventElapsedTime(&time, start, stop));
	CudaCheck(hipEventDestroy(start));
	CudaCheck(hipEventDestroy(stop));

	S_LOG_TRACE("     Sorting Execution: ", time, "ms");

	/*bvhHost->Build(faceAmount, mortonCodes, faces, vertices);
	CudaCheck(hipMemcpy(bvh, bvhHost, sizeof(BVH), hipMemcpyHostToDevice));*/
}

void Scene::Compile() {

	if (addList.size() > 0) {

		uint faceAmountPrevious = faceAmount;
		uint vertexAmountPrevious = vertexAmount;
		uint tetAmountPrevious = tetAmount;
		uint materialAmountPrevious = materialAmount;
		uint objectAmountPrevious = objectAmount;

		for (int i = 0; i < addList.size(); ++i) {

			tetAmount += addList[i]->tetAmount;
			faceAmount += addList[i]->faceAmount;
			vertexAmount += addList[i]->verticeAmount;
			materialAmount += addList[i]->materialAmount;
			++objectAmount;

		}



		//vertex resize
		if (vertexAmount > vertexAllocated) {
			Vertex* vertexTemp;

			vertexAllocated = glm::max(vertexAmount, uint(vertexAllocated*1.5f));
			CudaCheck(hipMalloc((void**)&vertexTemp, vertexAllocated * sizeof(Vertex)));

			if (vertices) {
				CudaCheck(hipMemcpy(vertexTemp, vertices, vertexAmountPrevious * sizeof(Vertex), hipMemcpyDeviceToDevice));
				CudaCheck(hipFree(vertices));
			}

			vertices = vertexTemp;
		}

		//face resize + morton codes
		if (faceAmount > faceAllocated) {
			Face* facesTemp;

			faceAllocated = glm::max(faceAmount, uint(faceAllocated*1.5f));
			CudaCheck(hipMalloc((void**)&facesTemp, faceAllocated * sizeof(Face)));

			if (faces) {
				CudaCheck(hipMemcpy(facesTemp, faces, faceAmountPrevious * sizeof(Face), hipMemcpyDeviceToDevice));
				CudaCheck(hipFree(faces));
			}

			faces = facesTemp;

			if (mortonCodes) {
				CudaCheck(hipFree(mortonCodes));
			}
			CudaCheck(hipMalloc((void**)&mortonCodes, faceAllocated * sizeof(uint64)));

		}

		//tet resize
		if (tetAmount > tetAllocated) {
			Tet* tetsTemp;

			tetAllocated = glm::max(tetAmount, uint(tetAllocated*1.5f));
			CudaCheck(hipMalloc((void**)&tetsTemp, tetAllocated * sizeof(Tet)));

			if (tets) {
				CudaCheck(hipMemcpy(tetsTemp, tets, tetAmountPrevious * sizeof(Tet), hipMemcpyDeviceToDevice));
				CudaCheck(hipFree(tets));
			}

			tets = tetsTemp;
		}

		//material resize
		if (materialAmount > materialAllocated) {
			Material* materialsTemp;

			materialAllocated = glm::max(materialAmount, uint(materialAllocated*1.5f));
			CudaCheck(hipMalloc((void**)&materialsTemp, materialAllocated * sizeof(Material)));

			if (materials) {
				CudaCheck(hipMemcpy(materialsTemp, materials, tetAmountPrevious * sizeof(Material), hipMemcpyDeviceToDevice));
				CudaCheck(hipFree(materials));
			}

			materials = materialsTemp;
		}

		//object resize
		if (objectAmount > objectAllocated) {
			MiniObject* objectsTemp;

			objectAllocated = glm::max(objectAmount, uint(objectAllocated*1.5f));
			CudaCheck(hipMalloc((void**)&objectsTemp, objectAllocated * sizeof(MiniObject)));

			if (objects) {
				CudaCheck(hipMemcpy(objectsTemp, objects, objectAmountPrevious * sizeof(MiniObject), hipMemcpyDeviceToDevice));
				CudaCheck(hipFree(objects));
			}

			objects = objectsTemp;
		}

		uint faceOffset = faceAmountPrevious;
		uint vertexOffset = vertexAmountPrevious;
		uint tetOffset = tetAmountPrevious;
		uint materialOffset = materialAmountPrevious;
		uint objectOffset = objectAmountPrevious;

		for (int i = 0; i < addList.size(); ++i) {

			//update the scene's bounding volume
			sceneBox.max = glm::max(sceneBox.max, addList[i]->box.max);
			sceneBox.min = glm::min(sceneBox.min, addList[i]->box.min);

			//create the modified host data to upload
			std::vector<Vertex> tempVertices(addList[i]->verticeAmount);
			std::vector<Face> tempFaces(addList[i]->faceAmount);
			std::vector<Tet> tempTets(addList[i]->tetAmount);
			std::vector<Material> tempMaterials(addList[i]->materialAmount);

			//create the minified object from the input object
			MiniObject tempObject(*addList[i]);

			uint maxIter = glm::max(addList[i]->materialAmount, glm::max(addList[i]->verticeAmount, glm::max(addList[i]->faceAmount, addList[i]->tetAmount)));

			for (uint t = 0; t < maxIter; ++t) {
				if (t < addList[i]->verticeAmount) {
					tempVertices[t] = addList[i]->vertices[t];
					tempVertices[t].object = objectOffset;
				}
				if (t < addList[i]->faceAmount) {
					tempFaces[t] = addList[i]->faces[t];
					tempFaces[t].material += materialOffset;
				}
				if (t < addList[i]->tetAmount) {
					tempTets[t] = addList[i]->tets[t];
					tempTets[t].material += materialOffset;
					tempTets[t].object = objectOffset;
				}
				if (t < addList[i]->materialAmount) {
					tempMaterials[t] = addList[i]->materials[t];
				}
			}


			//upload the data
			CudaCheck(hipMemcpy(vertices + vertexOffset, tempVertices.data(), tempVertices.size() * sizeof(Vertex), hipMemcpyHostToDevice));
			CudaCheck(hipMemcpy(faces + faceOffset, tempFaces.data(), tempFaces.size() * sizeof(Face), hipMemcpyHostToDevice));
			CudaCheck(hipMemcpy(tets + tetOffset, tempTets.data(), tempTets.size() * sizeof(Tet), hipMemcpyHostToDevice));
			CudaCheck(hipMemcpy(materials + materialOffset, tempMaterials.data(), tempMaterials.size() * sizeof(Material), hipMemcpyHostToDevice));
			CudaCheck(hipMemcpy(objects + objectOffset, &tempObject, sizeof(MiniObject), hipMemcpyHostToDevice));




			//update the offsets
			tetOffset += addList[i]->tetAmount;
			faceOffset += addList[i]->faceAmount;
			vertexOffset += addList[i]->verticeAmount;
			materialOffset += addList[i]->materialAmount;
			++objectOffset;

		}

		//clear the list
		addList.clear();
	}
}

//object pointer is host
void Scene::AddObject(Object* obj) {
	addList.push_back(obj);
}

void Scene::RemoveObject(Object* obj) {
	removeList.push_back(obj);
	S_LOG_WARNING("Removal of objects from a scene not yet implemented");
}
