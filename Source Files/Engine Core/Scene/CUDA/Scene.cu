#include "hip/hip_runtime.h"
#include "Scene.cuh"

#define RAY_BIAS_DISTANCE 0.0002f 
#define BVH_STACK_SIZE 64
#define DYNAMIC_FETCH_THRESHOLD 20          // If fewer than this active, fetch new rays

#include "Utility\CUDA\CUDAHelper.cuh"
#include "Utility\Logger.h"

#include "Algorithms\Morton Code\MortonCode.h"

#include <thrust/fill.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/remove.h>
#include <thrust/functional.h>

Scene::Scene()
{
	objectsSize = 0;
	allocatedObjects = 0;
	allocatedSize = 0;

	compiledSize = 0;
	newFaceAmount = 0;

	objectListHost.clear();
	objectsToRemove.clear();
	objectListDevice = nullptr;
	mortonCodes = nullptr;
	faceIds = nullptr;
	objIds = nullptr;
	objectBitSetup = nullptr;
	//give the addresses for the data
	bvhHost = new BVH(&faceIds, &mortonCodes);
	CudaCheck(hipMalloc((void **)&bvhDevice, sizeof(BVH)));

	Sky* skyHost = new Sky("Starmap.png");
	CudaCheck(hipMalloc((void **)&sky, sizeof(Sky)));
	CudaCheck(hipMemcpy(sky, skyHost, sizeof(Sky), hipMemcpyHostToDevice));

	CudaCheck(hipMalloc((void **)&sceneBoxDevice, sizeof(BoundingBox)));
}


Scene::~Scene()
{

	CudaCheck(hipFree(objectBitSetup)); // hold a true for the first indice of each object
	CudaCheck(hipFree(objIds)); //points to the object
	CudaCheck(hipFree(faceIds));
	CudaCheck(hipFree(mortonCodes));

	//Variables concerning object storage

	CudaCheck(hipFree(objectListDevice));
	CudaCheck(hipFree(objectRemoval));

	delete bvhHost;

	CudaCheck(hipFree(bvhDevice));
	CudaCheck(hipFree(sky));

}


struct is_scheduled
{
	__host__ __device__
		bool operator()(const Object* x)
	{
		return (x->requestRemoval);
	}
};


__global__ void FillBool(const uint n, bool* jobs, bool* fjobs, Face** faces, uint* objIds, Object** objects) {


	uint index = getGlobalIdx_1D_1D();


	if (index < n) {
		if (objects[objIds[index] - 1]->requestRemoval) {
			jobs[index] = true;
		}
		else {
			jobs[index] = false;
		}
		if (faces[index]->objectPointer->requestRemoval) {
			fjobs[index] = true;
		}
		else {
			fjobs[index] = false;
		}
	}
}

__global__ void GetFace(const uint n, uint* objIds, Object** objects, Face** faces, const uint offset) {


	uint index = getGlobalIdx_1D_1D();


	if (index >= n) {
		return;
	}

 	Object* obj = objects[objIds[offset + index] - 1];
	faces[offset + index] = obj->faces + (index - obj->localSceneIndex);
	faces[offset + index]->objectPointer = obj;

}


//add all new objects into the scene's arrays
__host__ bool Scene::Compile() {

	uint amountToRemove = objectsToRemove.size();

	if (newFaceAmount > 0 || amountToRemove > 0) {

		//bitSetup only has the first element of each object flagged
		//this extends that length and copies the previous results as well

		uint newSize = compiledSize + newFaceAmount;
		uint indicesToRemove = 0;
		uint removedOffset = 0;
		if (amountToRemove > 0) {

			bool* markers;
			bool* faceMarkers;
			CudaCheck(hipMalloc((void**)&markers, compiledSize * sizeof(bool)));
			CudaCheck(hipMalloc((void**)&faceMarkers, compiledSize * sizeof(bool)));

			thrust::device_ptr<bool> tempPtr = thrust::device_pointer_cast(markers);
			thrust::device_ptr<bool> faceTempPtr = thrust::device_pointer_cast(faceMarkers);


			//variables from the scene to the kernal

			uint blockSize = 64;
			uint gridSize = (compiledSize + blockSize - 1) / blockSize;

			//fill the mask with 1s or 0s
			FillBool << <gridSize, blockSize >> > (compiledSize, markers, faceMarkers, faceIds, objIds, objectListDevice);
			CudaCheck(hipPeekAtLastError());
			CudaCheck(hipDeviceSynchronize());


			//remove the requested
			thrust::device_ptr<bool> bitPtr = thrust::device_pointer_cast(objectBitSetup);

			thrust::device_ptr<bool> newEnd = thrust::remove_if(bitPtr, bitPtr + compiledSize, tempPtr, thrust::identity<bool>());
			CudaCheck(hipDeviceSynchronize());

			indicesToRemove = bitPtr + compiledSize - newEnd;
			newSize = newSize - indicesToRemove;
			//objpointers
			thrust::device_ptr<uint> objPtr = thrust::device_pointer_cast(objIds);

			thrust::remove_if(objPtr, objPtr + compiledSize, tempPtr, thrust::identity<bool>());
			CudaCheck(hipDeviceSynchronize());

			//faces
			thrust::device_ptr<Face*> facePtr = thrust::device_pointer_cast(faceIds);

			thrust::remove_if(facePtr, facePtr + compiledSize, faceTempPtr, thrust::identity<bool>());
			CudaCheck(hipDeviceSynchronize());

			//actual object list
			thrust::device_ptr<Object*> objectsPtr = thrust::device_pointer_cast(objectListDevice);

			thrust::remove_if(objectsPtr, objectsPtr + objectsSize, is_scheduled());

			CudaCheck(hipDeviceSynchronize());
			CudaCheck(hipFree(markers));
			CudaCheck(hipFree(faceMarkers));
		}

		if (newFaceAmount > 0) {

			if (allocatedSize < newSize) {
				Face** faceTemp;
				uint* objTemp;
				bool* objectBitSetupTemp;

				allocatedSize = glm::max(uint(allocatedSize * 1.5f), newSize);

				CudaCheck(hipMalloc((void**)&faceTemp, allocatedSize * sizeof(Face*)));
				CudaCheck(hipMalloc((void**)&objTemp, allocatedSize * sizeof(uint)));
				CudaCheck(hipMalloc((void**)&objectBitSetupTemp, allocatedSize * sizeof(bool)));

				if (mortonCodes) {
					CudaCheck(hipFree(mortonCodes));
				}
				CudaCheck(hipMalloc((void**)&mortonCodes, allocatedSize * sizeof(uint64)));

				if (faceIds) {
					CudaCheck(hipMemcpy(faceTemp, faceIds, compiledSize * sizeof(Face*), hipMemcpyDeviceToDevice));
					CudaCheck(hipFree(faceIds));
				}
				faceIds = faceTemp;

				if (objIds) {
					CudaCheck(hipMemcpy(objTemp, objIds, compiledSize * sizeof(uint), hipMemcpyDeviceToDevice));
					CudaCheck(hipFree(objIds));
				}
				objIds = objTemp;

				if (objectBitSetup) {
					CudaCheck(hipMemcpy(objectBitSetupTemp, objectBitSetup, compiledSize * sizeof(bool), hipMemcpyDeviceToDevice));
					CudaCheck(hipFree(objectBitSetup));
				}
				objectBitSetup = objectBitSetupTemp;

			}

			CudaCheck(hipDeviceSynchronize());
			removedOffset = compiledSize - indicesToRemove;
			//for each new object, (all at the end of the array) fill with falses.
			thrust::device_ptr<bool> bitPtr = thrust::device_pointer_cast(objectBitSetup);
			thrust::fill(bitPtr + removedOffset, bitPtr + newSize, (bool)false);

			CudaCheck(hipDeviceSynchronize());

		}


		CudaCheck(hipDeviceSynchronize());

		//flag the first and setup state of life (only time iteration through objects should be done)
		uint l = 0;
		for (uint i = 0; i < objectsSize; i++) {
			if (!objectListHost[i]->ready) {
				CudaCheck(hipMemset(objectBitSetup + l, true, sizeof(bool)));
				objectListHost[i]->ready = true;
			}
			
			objectListHost[i]->localSceneIndex = l;

			Object** objHolderHost = new Object*[1];

			Object* objDevice;
			CudaCheck(hipMalloc((void**)&objDevice, sizeof(Object)));
			CudaCheck(hipMemcpy(objDevice, objectListHost[i], sizeof(Object), hipMemcpyHostToDevice));

			objHolderHost[0] = objDevice;
			CudaCheck(hipMemcpy(objectListDevice + i, &objHolderHost[0], sizeof(Object*), hipMemcpyHostToDevice));
			l += objectListHost[i]->faceAmount;
		}

		if (newFaceAmount > 0) {

			thrust::device_ptr<bool> bitPtr = thrust::device_pointer_cast(objectBitSetup);
			thrust::device_ptr<uint> objPtr = thrust::device_pointer_cast(objIds);
			CudaCheck(hipDeviceSynchronize());

			thrust::inclusive_scan(bitPtr, bitPtr + newSize, objPtr);
			CudaCheck(hipDeviceSynchronize());


			uint blockSize = 64;
			uint gridSize = ((newSize - removedOffset) + blockSize - 1) / blockSize;

			GetFace << <gridSize, blockSize >> > (newSize - removedOffset, objIds, objectListDevice, faceIds, removedOffset);
			CudaCheck(hipPeekAtLastError());
			CudaCheck(hipDeviceSynchronize());

		}
		CudaCheck(hipDeviceSynchronize());

		//change the indice count of the scene
		compiledSize = newSize;
		newFaceAmount = 0;
		objectsToRemove.clear();
		return true;

	}
	else {
		return false;
	}


}


__host__ void Scene::Build(float deltaTime) {

	bool b = Compile();

	//calculate the morton code for each triangle

	uint blockSize = 64;
	uint gridSize = (compiledSize + blockSize - 1) / blockSize;

	CudaCheck(hipDeviceSynchronize());

	MortonCode::Compute << <gridSize, blockSize >> > (compiledSize, mortonCodes, faceIds, objectListDevice, sceneBoxDevice);


	CudaCheck(hipPeekAtLastError());
	CudaCheck(hipDeviceSynchronize());
	thrust::device_ptr<uint64_t> keys(mortonCodes);
	thrust::device_ptr<Face*> values(faceIds);


	CudaCheck(hipDeviceSynchronize());

	hipEvent_t start, stop;
	float time;
	CudaCheck(hipEventCreate(&start));
	CudaCheck(hipEventCreate(&stop));
	CudaCheck(hipEventRecord(start, 0));

	CudaCheck(hipDeviceSynchronize());

	thrust::sort_by_key(keys, keys + compiledSize, values);

	CudaCheck(hipDeviceSynchronize());

	CudaCheck(hipEventRecord(stop, 0));
	CudaCheck(hipEventSynchronize(stop));
	CudaCheck(hipEventElapsedTime(&time, start, stop));
	CudaCheck(hipEventDestroy(start));
	CudaCheck(hipEventDestroy(stop));

	S_LOG_TRACE("     Sorting Execution: ", time, "ms");

	bvhHost->Build(compiledSize);
	CudaCheck(hipMemcpy(bvhDevice, bvhHost, sizeof(BVH), hipMemcpyHostToDevice));


}

//object pointer is host
__host__ uint Scene::AddObject(Object*& obj) {

	//if the size of objects stores increases, double the available size pool;
	if (objectsSize == allocatedObjects) {

		Object** objectsTemp;
		allocatedObjects *= 2;

		if (allocatedObjects == 0) {
			allocatedObjects = 1;
		}

		CudaCheck(hipMalloc((void**)&objectsTemp, allocatedObjects * sizeof(Object*)));

		if (objectListDevice) {
			CudaCheck(hipMemcpy(objectsTemp, objectListDevice, objectsSize * sizeof(Object*), hipMemcpyDeviceToDevice));
			CudaCheck(hipFree(objectListDevice));
		}

		objectListDevice = objectsTemp;
	}


	//update the scene's bounding volume

	sceneBoxHost.max = glm::max(sceneBoxHost.max, obj->box.max);
	sceneBoxHost.min = glm::min(sceneBoxHost.min, obj->box.min);

	CudaCheck(hipMemcpy(sceneBoxDevice, &sceneBoxHost, sizeof(BoundingBox), hipMemcpyHostToDevice));

	objectListHost.push_back(obj);

	//add the reference as the new object and increase the object count by 1
	Object** objHolderHost = new Object*[1];

	Object* objDevice;
	CudaCheck(hipMalloc((void**)&objDevice, sizeof(Object)));
	CudaCheck(hipMemcpy(objDevice, obj, sizeof(Object), hipMemcpyHostToDevice));

	objHolderHost[0] = objDevice;
	CudaCheck(hipMemcpy(objectListDevice + objectsSize, &objHolderHost[0], sizeof(Object*), hipMemcpyHostToDevice));
	objectsSize++;
	newFaceAmount += obj->faceAmount;
	delete objHolderHost;

	return 0;
}

__host__ bool Scene::RemoveObject(const uint& tag) {
	objectRemoval[tag] = true;
	return true;
}
