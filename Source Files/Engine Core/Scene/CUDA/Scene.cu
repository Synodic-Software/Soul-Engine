#include "hip/hip_runtime.h"
#include "Scene.cuh"

#define RAY_BIAS_DISTANCE 0.0002f 
#define BVH_STACK_SIZE 64
#define DYNAMIC_FETCH_THRESHOLD 20          // If fewer than this active, fetch new rays

#include "Utility\CUDA\CUDAHelper.cuh"
#include "Utility\Logger.h"

#include "Algorithms\Morton Code\MortonCode.h"

#include <thrust/fill.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/remove.h>
#include <thrust/functional.h>

Scene::Scene()
{

	objects = nullptr;
	mortonCodes = nullptr;
	faces = nullptr;
	vertices = nullptr;
	materials = nullptr;
	tets = nullptr;

	faceAmount = 0;
	vertexAmount = 0;
	tetAmount = 0;
	materialAmount = 0;
	objectAmount = 0;

	faceAllocated = 0;
	vertexAllocated = 0;
	tetAllocated = 0;
	materialAllocated = 0;
	objectAllocated = 0;

	//
	CudaCheck(hipMalloc((void **)&bvhData, sizeof(BVHData)));

	skyHost = new Sky("Starmap.png");
	CudaCheck(hipMalloc((void **)&sky, sizeof(Sky)));
	CudaCheck(hipMemcpy(sky, skyHost, sizeof(Sky), hipMemcpyHostToDevice));
}


Scene::~Scene()
{

	CudaCheck(hipFree(mortonCodes));
	CudaCheck(hipFree(faces));
	CudaCheck(hipFree(vertices));
	CudaCheck(hipFree(tets));
	CudaCheck(hipFree(materials));
	CudaCheck(hipFree(objects));

	delete skyHost;

	CudaCheck(hipFree(bvhData));
	CudaCheck(hipFree(sky));

}

__host__ void Scene::Build(float deltaTime) {

	Compile();

	if (faceAmount > 0) {
		//calculate the morton code for each triangle
		uint blockSize = 64;
		uint gridSize = (faceAmount + blockSize - 1) / blockSize;


		MortonCode::Compute << <gridSize, blockSize >> > (faceAmount, mortonCodes, faces, vertices, sceneBox);

		CudaCheck(hipPeekAtLastError());
		CudaCheck(hipDeviceSynchronize());

		thrust::device_ptr<uint64_t> keys(mortonCodes);
		thrust::device_ptr<Face> values(faces);

		thrust::sort_by_key(keys, keys + faceAmount, values);

	}

	bvhHost.Build(faceAmount, bvhData, mortonCodes, faces, vertices);

}

void Scene::Compile() {

	if (addList.size() > 0) {

		uint faceAmountPrevious = faceAmount;
		uint vertexAmountPrevious = vertexAmount;
		uint tetAmountPrevious = tetAmount;
		uint materialAmountPrevious = materialAmount;
		uint objectAmountPrevious = objectAmount;

		for (int i = 0; i < addList.size(); ++i) {

			tetAmount += addList[i].second->tetAmount;
			faceAmount += addList[i].second->faceAmount;
			vertexAmount += addList[i].second->verticeAmount;
			materialAmount += addList[i].second->materialAmount;
			++objectAmount;

		}

		for (int i = 0; i < cameraList.size(); ++i) {
			++faceAmount;
			++vertexAmount;
			++objectAmount;
		}

		//vertex resize
		if (vertexAmount > vertexAllocated) {
			Vertex* vertexTemp;

			vertexAllocated = glm::max(vertexAmount, uint(vertexAllocated*1.5f));
			CudaCheck(hipMalloc((void**)&vertexTemp, vertexAllocated * sizeof(Vertex)));

			if (vertices) {
				CudaCheck(hipMemcpy(vertexTemp, vertices, vertexAmountPrevious * sizeof(Vertex), hipMemcpyDeviceToDevice));
				CudaCheck(hipFree(vertices));
			}

			vertices = vertexTemp;
		}

		//face resize + morton codes
		if (faceAmount > faceAllocated) {
			Face* facesTemp;

			faceAllocated = glm::max(faceAmount, uint(faceAllocated*1.5f));
			CudaCheck(hipMalloc((void**)&facesTemp, faceAllocated * sizeof(Face)));

			if (faces) {
				CudaCheck(hipMemcpy(facesTemp, faces, faceAmountPrevious * sizeof(Face), hipMemcpyDeviceToDevice));
				CudaCheck(hipFree(faces));
			}

			faces = facesTemp;

			if (mortonCodes) {
				CudaCheck(hipFree(mortonCodes));
			}
			CudaCheck(hipMalloc((void**)&mortonCodes, faceAllocated * sizeof(uint64)));

		}

		//tet resize
		if (tetAmount > tetAllocated) {
			Tet* tetsTemp;

			tetAllocated = glm::max(tetAmount, uint(tetAllocated*1.5f));
			CudaCheck(hipMalloc((void**)&tetsTemp, tetAllocated * sizeof(Tet)));

			if (tets) {
				CudaCheck(hipMemcpy(tetsTemp, tets, tetAmountPrevious * sizeof(Tet), hipMemcpyDeviceToDevice));
				CudaCheck(hipFree(tets));
			}

			tets = tetsTemp;
		}

		//material resize
		if (materialAmount > materialAllocated) {
			Material* materialsTemp;

			materialAllocated = glm::max(materialAmount, uint(materialAllocated*1.5f));
			CudaCheck(hipMalloc((void**)&materialsTemp, materialAllocated * sizeof(Material)));

			if (materials) {
				CudaCheck(hipMemcpy(materialsTemp, materials, tetAmountPrevious * sizeof(Material), hipMemcpyDeviceToDevice));
				CudaCheck(hipFree(materials));
			}

			materials = materialsTemp;
		}

		//object resize
		if (objectAmount > objectAllocated) {
			MiniObject* objectsTemp;

			objectAllocated = glm::max(objectAmount, uint(objectAllocated*1.5f));
			CudaCheck(hipMalloc((void**)&objectsTemp, objectAllocated * sizeof(MiniObject)));

			if (objects) {
				CudaCheck(hipMemcpy(objectsTemp, objects, objectAmountPrevious * sizeof(MiniObject), hipMemcpyDeviceToDevice));
				CudaCheck(hipFree(objects));
			}

			objects = objectsTemp;
		}

		uint faceOffset = faceAmountPrevious;
		uint vertexOffset = vertexAmountPrevious;
		uint tetOffset = tetAmountPrevious;
		uint materialOffset = materialAmountPrevious;
		uint objectOffset = objectAmountPrevious;

		for (int i = 0; i < addList.size(); ++i) {

			//update the scene's bounding volume
			sceneBox.max = glm::max(sceneBox.max, addList[i].second->box.max);
			sceneBox.min = glm::min(sceneBox.min, addList[i].second->box.min);

			//create the modified host data to upload
			std::vector<Vertex> tempVertices(addList[i].second->verticeAmount);
			std::vector<Face> tempFaces(addList[i].second->faceAmount);
			std::vector<Tet> tempTets(addList[i].second->tetAmount);
			std::vector<Material> tempMaterials(addList[i].second->materialAmount);

			//create the minified object from the input object
			MiniObject tempObject(*addList[i].second);

			uint maxIter = glm::max(addList[i].second->materialAmount, glm::max(addList[i].second->verticeAmount, glm::max(addList[i].second->faceAmount, addList[i].second->tetAmount)));

			for (uint t = 0; t < maxIter; ++t) {
				if (t < addList[i].second->verticeAmount) {
					tempVertices[t] = addList[i].second->vertices[t];
					glm::vec4 pos = glm::vec4(tempVertices[t].position.x, tempVertices[t].position.y, tempVertices[t].position.z, 1.0f);
					pos = addList[i].first*pos;
					tempVertices[t].position = glm::vec3(pos.x, pos.y, pos.z);
					tempVertices[t].object = objectOffset;
				}
				if (t < addList[i].second->faceAmount) {
					tempFaces[t] = addList[i].second->faces[t];

					tempFaces[t].indices.x += vertexOffset;
					tempFaces[t].indices.y += vertexOffset;
					tempFaces[t].indices.z += vertexOffset;

					tempFaces[t].material += materialOffset;
				}
				if (t < addList[i].second->tetAmount) {
					tempTets[t] = addList[i].second->tets[t];
					tempTets[t].material += materialOffset;
					tempTets[t].object = objectOffset;
				}
				if (t < addList[i].second->materialAmount) {
					tempMaterials[t] = addList[i].second->materials[t];
				}
			}


			//upload the data
			CudaCheck(hipMemcpy(vertices + vertexOffset, tempVertices.data(), tempVertices.size() * sizeof(Vertex), hipMemcpyHostToDevice));
			CudaCheck(hipMemcpy(faces + faceOffset, tempFaces.data(), tempFaces.size() * sizeof(Face), hipMemcpyHostToDevice));
			CudaCheck(hipMemcpy(tets + tetOffset, tempTets.data(), tempTets.size() * sizeof(Tet), hipMemcpyHostToDevice));
			CudaCheck(hipMemcpy(materials + materialOffset, tempMaterials.data(), tempMaterials.size() * sizeof(Material), hipMemcpyHostToDevice));
			CudaCheck(hipMemcpy(objects + objectOffset, &tempObject, sizeof(MiniObject), hipMemcpyHostToDevice));


			//update the offsets
			tetOffset += addList[i].second->tetAmount;
			faceOffset += addList[i].second->faceAmount;
			vertexOffset += addList[i].second->verticeAmount;
			materialOffset += addList[i].second->materialAmount;
			++objectOffset;

		}

		for (int i = 0; i < cameraList.size(); ++i) {

			MiniObject obj;

			Vertex vertex;

			glm::vec4 pos = glm::vec4(
				cameraList[i].second->Position().x, 
				cameraList[i].second->Position().y, 
				cameraList[i].second->Position().z, 1.0f);

			pos = cameraList[i].first*pos;
			vertex.position = glm::vec3(pos.x, pos.y, pos.z);
			vertex.object = objectOffset;

			Face face;
			face.material = 0;
			face.indices.x = vertexOffset;
			face.indices.y = vertexOffset;
			face.indices.z = vertexOffset;

			cameraList[i].second->currentVert = vertex;
			cameraList[i].second->devicePos = vertices+vertexOffset;

			CudaCheck(hipMemcpy(vertices + vertexOffset, &vertex, sizeof(Vertex), hipMemcpyHostToDevice));
			CudaCheck(hipMemcpy(faces + faceOffset, &face, sizeof(Face), hipMemcpyHostToDevice));
			CudaCheck(hipMemcpy(objects + objectOffset, &obj, sizeof(MiniObject), hipMemcpyHostToDevice));

			++vertexOffset;
			++faceOffset;
			++objectOffset;

		}

		//clear the list
		addList.clear();
		cameraList.clear();
	}
}

//object pointer is host
void Scene::AddObject(std::vector<SceneNode> matrix, Object* obj) {
	addList.push_back(std::make_pair(matrix, obj));
}

void Scene::AddCamera(std::vector<SceneNode> matrix, Camera* camera) {
	cameraList.push_back(std::make_pair(matrix, camera));
}

void Scene::OverwriteSceneGraph(SceneNode* root) {
	sceneGraph = root;
}

void Scene::RemoveObject(Object* obj) {
	removeList.push_back(obj);
	S_LOG_WARNING("Removal of objects from a scene not yet implemented");
}
