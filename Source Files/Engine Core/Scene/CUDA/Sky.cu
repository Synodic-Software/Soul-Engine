#include "hip/hip_runtime.h"
#include "Sky.cuh"

Sky::Sky(std::string texName) {

	//image = new Image();

	//CudaCheck(hipDeviceSynchronize());
	//image->LoadFromFile(texName.c_str(),true,false);
	//CudaCheck(hipDeviceSynchronize());
}

__device__ glm::vec3 Sky::ExtractColour(const glm::vec3& direction){

	/*float theta = 0.5f + atan2f(direction.z, direction.x)/(2 * PI);
	float gamma = 0.5f - asinf(direction.y)/ PI;
	float4 col = tex2D<float4>(image->texObj, theta, gamma );*/


	float normalized = direction.y / 2.0f + 0.5f;
	return glm::mix(glm::vec3(255.0f / 255.0f, 255.0f / 255.0f, 255.0f / 255.0f),glm::vec3(135 / 255.0f, 200 / 255.0f, 240 / 255.0f),normalized/2.0f+0.5f)*0.5f;
}
