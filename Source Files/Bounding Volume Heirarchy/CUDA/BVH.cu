#include "hip/hip_runtime.h"
#include "BVH.cuh"
#include "Utility\CUDA\CUDAHelper.cuh"
#include "Utility/Logger.h"

BVH::BVH( Face*** datan, uint64** mortonCodesn){
	data = datan;
	mortonCodes = mortonCodesn;
	currentSize = 0;
	allocatedSize = 0;
}

// Returns the highest differing bit of i and i+1
__device__ uint HighestBit(uint i, uint64* morton)
{
	return morton[i] ^ morton[i + 1];
}

__global__ void BuildTree(const uint n, Node* nodes, Face** data, uint64* mortonCodes, const uint leafOffset, BVH* bvh)
{
	uint index = getGlobalIdx_1D_1D();
	if (index >= n)
		return;

	Node* currentNode = nodes + (leafOffset + index);

	while (true){
		// Allow only one thread to process a node
		if (atomicAdd(&(currentNode->atomic), 1) != 1)
			return;

		// Set bounding box if the node is no leaf
		if (currentNode - nodes < leafOffset)
		{
			currentNode->box.max = glm::max(currentNode->childLeft->box.max, currentNode->childRight->box.max);
			currentNode->box.min = glm::min(currentNode->childLeft->box.min, currentNode->childRight->box.min);
		}

		uint left = currentNode->rangeLeft;
		uint right = currentNode->rangeRight;

		if (left == 0 && right == leafOffset){
			bvh->root = currentNode;
			return;
		}

		Node* parent;
		if (left == 0 || (right < leafOffset && HighestBit(left - 1, mortonCodes) > HighestBit(right, mortonCodes)))
		{
			// parent = right, set parent left child and range to node
			parent = nodes + right;
			parent->childLeft = currentNode;
			parent->rangeLeft = left;

		}
		else
		{
			// parent = left -1, set parent right child and range to node
			parent = nodes + (left - 1);
			parent->childRight = currentNode;
			parent->rangeRight = right;
		}

		currentNode = parent;
	}
}
	

__global__ void Reset(const uint n,Node* nodes, Face** data, uint64* mortonCodes,const uint leafOffset)
{
	uint index = getGlobalIdx_1D_1D();
	if (index >= n)
		return;

	// Reset parameters for internal and leaf nodes here

	// Set ranges
	nodes[leafOffset + index].rangeLeft = index;
	nodes[leafOffset + index].rangeRight = index;
	nodes[leafOffset + index].atomic = 1; // To allow the next thread to process
	nodes[leafOffset + index].childLeft = NULL; // Second thread to process
	nodes[leafOffset + index].childRight = NULL; // Second thread to process
	if (index<leafOffset){
		/*nodes[index].rangeLeft = index;   //unneeded as all nodes are touched and updated
		nodes[index].rangeRight = index + 1;*/
		nodes[index].atomic = 0; // Second thread to process
		nodes[index].childLeft = &nodes[leafOffset + index]; // Second thread to process
		nodes[index].childRight = &nodes[leafOffset + index+1]; // Second thread to process
	}


	// Set triangles in leaf
	Face* face = *(data + index);
	nodes[leafOffset + index].faceID = face;

	// Expand bounds using min/max functions

	glm::vec3 max = face->objectPointer->vertices[face->indices.x].position;
	glm::vec3 min = face->objectPointer->vertices[face->indices.x].position;

	max = glm::max(face->objectPointer->vertices[face->indices.y].position, max);
	min = glm::min(face->objectPointer->vertices[face->indices.y].position, min);

	max = glm::max(face->objectPointer->vertices[face->indices.z].position, max);
	min = glm::min(face->objectPointer->vertices[face->indices.z].position, min);

	nodes[leafOffset + index].box.max = max;
	nodes[leafOffset + index].box.min = min;

	// Special case
	if (n == 1)
	{
		nodes[0].box = nodes[leafOffset+0].box;
		nodes[0].childLeft = &nodes[leafOffset+0];
	}
}

void BVH::Build(uint size){
	hipEvent_t start, stop;
	float time;
	CudaCheck(hipEventCreate(&start));
	CudaCheck(hipEventCreate(&stop));
	CudaCheck(hipEventRecord(start, 0));

	currentSize = size;
	if (currentSize > allocatedSize){

		Node* nodeTemp;

		allocatedSize = glm::max(uint(allocatedSize * 1.5f), (currentSize * 2) - 1);


		CudaCheck(hipMallocManaged((void**)&nodeTemp, allocatedSize * sizeof(Node)));

		CudaCheck(hipFree(bvh));
		bvh = nodeTemp;
	}

	root = bvh;

	uint blockSize = 64;
	uint gridSize = (currentSize + blockSize - 1) / blockSize;

	CudaCheck(hipDeviceSynchronize());

	Reset << <gridSize, blockSize >> >(currentSize, bvh, *data, *mortonCodes, currentSize - 1);
	CudaCheck(hipPeekAtLastError());
	CudaCheck(hipDeviceSynchronize());

	BuildTree << <gridSize, blockSize >> >(currentSize, bvh, *data, *mortonCodes, currentSize - 1,this);

	CudaCheck(hipPeekAtLastError());
	CudaCheck(hipDeviceSynchronize());

	CudaCheck(hipEventRecord(stop, 0));
	CudaCheck(hipEventSynchronize(stop));
	CudaCheck(hipEventElapsedTime(&time, start, stop));
	CudaCheck(hipEventDestroy(start));
	CudaCheck(hipEventDestroy(stop));

	S_LOG_TRACE( "     BVH Creation Execution: " , time , "ms" );
}