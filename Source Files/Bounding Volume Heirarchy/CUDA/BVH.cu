#include "hip/hip_runtime.h"
#include "BVH.cuh"
#include "Utility\CUDA\CUDAHelper.cuh"
#include "Utility/Logger.h"

BVH::BVH() {

	allocatedSize = 0;
	bvhDataHost.currentSize = 0;
	bvh = nullptr;

}

BVH::~BVH() {

	if (bvh) {
		CudaCheck(hipFree(bvh));
	}

}

// Returns the highest differing bit of i and i+1
__device__ uint HighestBit(MiniObject& objThis, uint64 mortonThis, MiniObject& objNext, uint64 mortonNext)
{
	uint zeroCount = __clzll(mortonThis^mortonNext);
	uint bitCount = 64 - zeroCount;
	for (int t = glm::max(objThis.tSize, objNext.tSize); t >= 0; --t) {
		if (t < objThis.tSize) {

		}
		if (t < objNext.tSize) {

		}
	}
	return bitCount;
}

__global__ void BuildTree(const uint n, BVHData* data, Node* nodes, Face* faces, Vertex* vertices, MiniObject* objects, const uint leafOffset)
{
	uint index = getGlobalIdx_1D_1D();
	if (index >= n)
		return;

	Node* currentNode = nodes + (leafOffset + index);

	while (true) {
		// Allow only one thread to process a node
		if (atomicAdd(&(currentNode->atomic), 1) != 1)
			return;

		// Set bounding box if the node is not a leaf
		if (currentNode - nodes < leafOffset)
		{
			currentNode->box.max = glm::max(currentNode->childLeft->box.max, currentNode->childRight->box.max);
			currentNode->box.min = glm::min(currentNode->childLeft->box.min, currentNode->childRight->box.min);
		}

		uint left = currentNode->rangeLeft;
		uint right = currentNode->rangeRight;

		if (left == 0 && right == leafOffset) {
			data->root = currentNode;
			return;
		}

		Node* parent;
		Face a = faces[nodes[leafOffset + left - 1].faceID];
		Face b = faces[nodes[leafOffset + left].faceID];
		Face c = faces[nodes[leafOffset + right].faceID];
		Face d = faces[nodes[leafOffset + right + 1].faceID];
		if (left == 0 || (right < leafOffset && HighestBit(
			objects[vertices[a.indices.x].object], a.mortonCode,
			objects[vertices[b.indices.x].object], b.mortonCode) >
			HighestBit(
				objects[vertices[c.indices.x].object], c.mortonCode,
				objects[vertices[d.indices.x].object], d.mortonCode)))
		{
			// parent = right, set parent left child and range to node
			parent = nodes + right;
			parent->childLeft = currentNode;
			parent->rangeLeft = left;

		}
		else
		{
			// parent = left -1, set parent right child and range to node
			parent = nodes + (left - 1);
			parent->childRight = currentNode;
			parent->rangeRight = right;
		}

		currentNode = parent;
	}
}


__global__ void Reset(const uint n, Node* nodes, Face* faces, Vertex* vertices, const uint leafOffset)
{
	uint index = getGlobalIdx_1D_1D();

	if (index >= n) {
		return;
	}

	Node temp;

	// Set ranges
	temp.rangeLeft = index;
	temp.rangeRight = index;
	temp.atomic = 1; // To allow the next thread to process
	temp.childLeft = nullptr;
	temp.childRight = nullptr;
	temp.transformLeft = glm::mat4();
	temp.transformRight = glm::mat4();

	if (index < leafOffset) {
		Node tempF;

		tempF.atomic = 0;
		tempF.childLeft = nodes + leafOffset + index;
		tempF.childRight = nodes + leafOffset + index + 1;

		nodes[index] = tempF;
	}

	// Set triangles in leaf
	temp.faceID = index;

	glm::uvec3 ind = faces[index].indices;

	// Expand bounds using min/max functions
	glm::vec3 pos0 = vertices[ind.x].position;
	glm::vec3 max = pos0;
	glm::vec3 min = pos0;

	glm::vec3 pos1 = vertices[ind.y].position;
	max = glm::max(pos1, max);
	min = glm::min(pos1, min);

	glm::vec3 pos2 = vertices[ind.z].position;
	max = glm::max(pos2, max);
	min = glm::min(pos2, min);

	temp.box.max = max;
	temp.box.min = min;

	nodes[leafOffset + index] = temp;

	// Special case
	if (n == 1)
	{
		nodes[0].box = nodes[leafOffset + 0].box;
		nodes[0].childLeft = nodes + leafOffset + 0;
	}
}

void BVH::Build(uint size, BVHData*& data, Face * faces, Vertex * vertices, MiniObject* objects) {

	if (size > 0) {
		if (size > allocatedSize) {

			allocatedSize = glm::max(uint(allocatedSize * 1.5f), (size * 2) - 1);

			if (bvh) {
				CudaCheck(hipFree(bvh));
			}

			CudaCheck(hipMalloc((void**)&bvh, allocatedSize * sizeof(Node)));
			bvhDataHost.bvh = bvh;
		}

		bvhDataHost.currentSize = size;
		CudaCheck(hipMemcpy(data, &bvhDataHost, sizeof(BVHData), hipMemcpyHostToDevice));

		uint blockSize = 64;
		uint gridSize = (size + blockSize - 1) / blockSize;

		CudaCheck(hipDeviceSynchronize());

		Reset << <gridSize, blockSize >> > (size, bvh, faces, vertices, size - 1);
		CudaCheck(hipPeekAtLastError());
		CudaCheck(hipDeviceSynchronize());

		BuildTree << <gridSize, blockSize >> > (size, data, bvh, faces, vertices, objects, size - 1);
		CudaCheck(hipPeekAtLastError());
		CudaCheck(hipDeviceSynchronize());
	}

}