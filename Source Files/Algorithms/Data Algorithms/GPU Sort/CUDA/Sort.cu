//#include "Sort.cuh"

#include <thrust/sort.h>
#include <thrust/device_ptr.h>

#include "Utility/CUDA/CudaHelper.cuh"
#include "Metrics.h"

namespace Sort {

	uint* Calculate(uint size, uint64* keys, uint* values) {

		uint64* keysDevice;
		uint* valuesDevice;

		//create device space
		CudaCheck(hipMalloc((void **)&keysDevice, size * sizeof(uint64)));
		CudaCheck(hipMalloc((void **)&valuesDevice, size * sizeof(uint)));

		//tranfer memory
		CudaCheck(hipMemcpy(keysDevice, keys, size * sizeof(uint64), hipMemcpyHostToDevice));
		CudaCheck(hipMemcpy(valuesDevice, values, size * sizeof(uint), hipMemcpyHostToDevice));

		//sort with thrust
		thrust::device_ptr<uint64> key_ptr = thrust::device_pointer_cast(keysDevice);
		thrust::device_ptr<uint> value_ptr = thrust::device_pointer_cast(valuesDevice);

		thrust::sort_by_key(key_ptr, key_ptr + size, value_ptr);

		CudaCheck(hipFree(keysDevice));

		return valuesDevice;
	}

}