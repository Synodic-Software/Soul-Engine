#include "Utility\CUDA\CUDAManaged.cuh"
#include <hip/hip_runtime.h>
#include "Utility\CUDA\CUDAHelper.cuh" 

void* Managed::operator new(size_t len){
	void *ptr;
	CudaCheck(hipMallocManaged((void**)&ptr, len));
	CudaCheck(hipDeviceSynchronize());
	return ptr;
}

void Managed::operator delete(void *ptr) {
	CudaCheck(hipDeviceSynchronize());
	CudaCheck(hipFree(ptr));
}

