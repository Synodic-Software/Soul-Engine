#include "hip/hip_runtime.h"
#include "RenderWidget.cuh"
#include "Utility\CUDA\CUDAHelper.cuh"

__global__ void IntegrateKernal(const uint n, glm::vec4* A, glm::vec4* B, const uint counter) {


	uint index = getGlobalIdx_1D_1D();

	if (index < n) {
		B[index] = glm::mix(B[index], A[index], 1.0f / counter);
		A[index] = B[index];
	}
}


__host__ void Integrate(RayJob* RenderJob, glm::vec4* A, glm::vec4* B, const uint counter) {

	uint n = RenderJob->rayAmount;
	uint blockSize = 64;
	uint gridSize = (n + blockSize - 1) / blockSize;

	IntegrateKernal << <gridSize, blockSize >> >(n, A, B, counter);
	CudaCheck(hipPeekAtLastError());
	CudaCheck(hipDeviceSynchronize());

}