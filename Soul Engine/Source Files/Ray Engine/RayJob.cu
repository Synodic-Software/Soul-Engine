#include "RayJob.cuh"

RayJob::RayJob(castType whatToGet, uint rayAmountN, uint newSamples, Camera* cameraN, bool isRecurringN){

	type = whatToGet;
	rayAmount = rayAmountN;
	rayBaseAmount = rayAmount;
	samples = newSamples;
	camera = cameraN;
	isRecurring = isRecurringN;
	nextRay = NULL;

	if (whatToGet != RayOBJECT_ID&&!RayCOLOUR_TO_TEXTURE){
		hipMallocManaged(&resultsF, rayBaseAmount);
		resultsI = NULL;
		resultsT = NULL;
	}
	else if (RayCOLOUR_TO_TEXTURE){

		

		resultsI = NULL;
		resultsF = NULL;
	}
	else{
		hipMallocManaged(&resultsI, rayBaseAmount);
		resultsF = NULL;
		resultsT = NULL;
	}
}