#include "hip/hip_runtime.h"
#include "RayJob.cuh"

__host__ RayJob::RayJob(rayType whatToGet, uint rayAmountN, uint newSamples, Camera* cameraN, uint numResultBuffersN){

	type = whatToGet;
	rayAmount = rayAmountN;
	rayBaseAmount = rayAmount;
	samples = newSamples;
	camera = cameraN;
	numResultBuffers = numResultBuffersN;

	CudaCheck(hipMallocManaged((void**)&results, numResultBuffers*sizeof(glm::vec4*)));
	for (int i = 0; i < numResultBuffers; i++){
		CudaCheck(hipMallocManaged((void**)&results[i], rayBaseAmount*sizeof(glm::vec4)));
	}

}

__host__ RayJob::~RayJob(){
	if (results != NULL){
		for (int i = 0; i < numResultBuffers; i++){
			if (results[i] != NULL){
				hipFree(results[i]);
			}
		}
		hipFree(results);
	}
}

//Returns a reference to a camera pointer. All the ray shooting information is stored here.
CUDA_FUNCTION Camera*& RayJob::GetCamera(){
	return camera;
}

//Returns the rayType of the job.
CUDA_FUNCTION rayType RayJob::RayType() const{
	return type;
}

//Returns the Ray max of the job as per its initialization params.
CUDA_FUNCTION uint RayJob::RayAmountMax() const{
	return rayBaseAmount;
}

//Returns the current rayAmount (modifiable)
CUDA_FUNCTION uint& RayJob::GetRayAmount() {
	return rayAmount;
}

//Returns the current sample per ray (modifiable)
CUDA_FUNCTION uint& RayJob::GetSampleAmount() {
	return samples;
}

//Returns the pointer to the results (modifiable)
CUDA_FUNCTION void*& RayJob::GetResultPointer(uint x){
	return results[x];
}

CUDA_FUNCTION void RayJob::SwapResults(uint a, uint b){
	void* temp = results[a];
	results[a] = results[b];
	results[b] = temp;
}
