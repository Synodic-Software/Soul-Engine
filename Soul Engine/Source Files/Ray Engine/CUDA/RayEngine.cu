#include "hip/hip_runtime.h"
#include "Ray Engine\RayEngine.cuh"

uint raySeed=0;

__device__ uint getGlobalIdx_1D_1D()
{
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__global__ void EngineExecute(uint n, RayJob& jobs, uint raySeed){
	uint index = getGlobalIdx_1D_1D();

	if (index < n){
		RayJob job = jobs;
		uint n = 0;

		while (jobs.nextRay != NULL && !(index < n + jobs.rayAmount)){
			n += job.rayAmount*job.samples;
			job = *job.nextRay;
		}

		uint localIndex = index - n;

		Ray ray = job.camera->SetupRay(index, n, raySeed);

		glm::vec2 fov= job.camera->FieldOfView();
		float aspectRatio = fov.x / fov.y;
		int screenX = (n*aspectRatio) / (aspectRatio + 1);
		int screenY = n / screenX;
		uint i = localIndex % screenX;
		uint j = localIndex / screenX;

		//calculate something


		if (job.type != RayOBJECT_ID&&!RayCOLOUR_TO_TEXTURE){
			job.GetResultFloat()[localIndex] = glm::vec3(0.5f, 0.5f, 0.5f);
		}
		else if (RayCOLOUR_TO_TEXTURE){
			job.GetResultBuffer()[localIndex] = glm::vec4(0.5f, 0.5f, 0.5f,1.0f);
		}
		else{
			job.GetResultInt()[localIndex] = 1;
		}
	}
}

__host__ void ProcessJobs(RayJob* jobs){
	raySeed++;

	if (jobs!=NULL){
	uint n = 0;

	RayJob* temp = jobs;
	n += temp->rayAmount;
	while (temp->nextRay != NULL){
		temp = temp->nextRay;
		n += temp->rayAmount*temp->samples;
	}

	if (n!=0){

		const int warpSize = 32;
		const int maxGridSize = 112; // this is 8 blocks per MP for a Telsa C2050

		int warpCount = (n / warpSize) + (((n % warpSize) == 0) ? 0 : 1);
		int warpPerBlock = glm::max(1, glm::min(4, warpCount));

		// For the cdiv kernel, the block size is allowed to grow to
		// four warps per block, and the block count becomes the warp count over four
		// or the GPU "fill" whichever is smaller
		int threadCount = warpSize * warpPerBlock;
		int blockCount = glm::min(maxGridSize, glm::max(1, warpCount / warpPerBlock));
		dim3 BlockDim = dim3(threadCount, 1, 1);
		dim3 GridDim = dim3(blockCount, 1, 1);


		//execute engine
		EngineExecute << <GridDim, BlockDim >> >(n, *jobs, raySeed);

	}
	}


}