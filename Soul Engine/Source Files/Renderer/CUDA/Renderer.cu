#include "hip/hip_runtime.h"
#include "Renderer.cuh"
#include "Utility\CUDA\CUDAHelper.cuh"

__global__ void IntegrateKernal(const uint n, RayJob* job, const uint counter){


	uint index = getGlobalIdx_1D_1D();

	if (index < n){
		((glm::vec4*)job->GetResultPointer(1))[index] = glm::mix(((glm::vec4*)job->GetResultPointer(1))[index], ((glm::vec4*)job->GetResultPointer(0))[index], 1.0f / counter);
		((glm::vec4*)job->GetResultPointer(0))[index] = ((glm::vec4*)job->GetResultPointer(1))[index];
	}
}


__host__ void Integrate(RayJob* RenderJob,const uint counter){
	//RenderJob->SwapResults(0, 1);

	uint n = RenderJob->GetRayAmount();
	uint blockSize = 64;
	uint gridSize = (n + blockSize - 1) / blockSize;

	hipEvent_t start, stop;
	float time;
	CudaCheck(hipEventCreate(&start));
	CudaCheck(hipEventCreate(&stop));
	CudaCheck(hipEventRecord(start, 0));

	IntegrateKernal << <gridSize, blockSize >> >(n, RenderJob, counter);
	CudaCheck(hipPeekAtLastError());
	CudaCheck(hipDeviceSynchronize());
	CudaCheck(hipEventRecord(stop, 0));
	CudaCheck(hipEventSynchronize(stop));
	CudaCheck(hipEventElapsedTime(&time, start, stop));
	CudaCheck(hipEventDestroy(start));
	CudaCheck(hipEventDestroy(stop));

	std::cout << "Colour Merge Execution: " << time << "ms" << std::endl;
}