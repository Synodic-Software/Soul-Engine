#include "hip/hip_runtime.h"
#include "Renderer.cuh"

__global__ void IntegrateKernal(const uint n, RayJob* job, const uint counter){


	uint index = getGlobalIdx_1D_1D();

	if (index < n){
		((glm::vec4*)job->GetResultPointer(1))[index] = glm::mix(((glm::vec4*)job->GetResultPointer(1))[index], ((glm::vec4*)job->GetResultPointer(0))[index], 1.0f / counter);
		((glm::vec4*)job->GetResultPointer(0))[index] = ((glm::vec4*)job->GetResultPointer(1))[index];
	}
}


__host__ void Integrate(RayJob* RenderJob,const uint counter){
	//RenderJob->SwapResults(0, 1);

	uint n = RenderJob->GetRayAmount();
	uint blockSize = 64;
	uint gridSize = (n + blockSize - 1) / blockSize;

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	IntegrateKernal << <gridSize, blockSize >> >(n, RenderJob, counter);

	CudaCheck(hipEventRecord(stop, 0));
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	std::cout << "Colour Merge Execution: " << time << "ms" << std::endl;
}