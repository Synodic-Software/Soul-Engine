#include "hip/hip_runtime.h"
#include "Renderer.cuh"

__global__ void IntegrateKernal(const uint n, RayJob* job, const uint counter){


	uint index = getGlobalIdx_1D_1D();

	if (index < n){

		((glm::vec4*)job->GetResultPointer(0))[index] = glm::mix(((glm::vec4*)job->GetResultPointer(0))[index], ((glm::vec4*)job->GetResultPointer(1))[index], 1.0f / counter);
	}
}


__host__ void Integrate(RayJob* RenderJob,const uint counter){
	RenderJob->SwapResults(0, 1);

	uint n = RenderJob->GetRayAmount();
	uint blockSize = 64;
	uint gridSize = (n + blockSize - 1) / blockSize;

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	IntegrateKernal << <gridSize, blockSize >> >(n,RenderJob,counter);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	std::cout << "RayClear Execution: " << time << "ms" << std::endl;
}