#include "hip/hip_runtime.h"
#include "Utility\CUDA\CUDADevices.cuh"
#include <>
#include <hip/hip_runtime_api.h>
#include "CudaHelper.cuh"
#include "Utility\Logger.h"

int deviceCount;
hipDeviceProp_t* deviceProp;

void Devices::ExtractDevices(){
	hipError_t error = hipGetDeviceCount(&deviceCount);

	if (deviceCount == 0)
	{
		Logger::Log(TRACE, "There are no available device(s) that support CUDA\n");
	}

	deviceProp = new hipDeviceProp_t[deviceCount];

	for (int dev = 0; dev < deviceCount; ++dev){

		CudaCheck(hipSetDevice(dev));
		CudaCheck(hipGetDeviceProperties(&deviceProp[dev], dev));

	}

	///REMEMBER CUDA_VISIBLE_DEVICS="0" in command line c++ settings

	CudaCheck(hipSetDevice(0));
}

int Devices::GetCoreCount(){
	int device;
	CudaCheck(hipGetDevice(&device));
	return _ConvertSMVer2Cores(deviceProp[device].major, deviceProp[device].minor) * deviceProp[device].multiProcessorCount;
}

int Devices::GetSMCount(){
	int device;
	CudaCheck(hipGetDevice(&device));
	return deviceProp[device].multiProcessorCount;
}
//
//int Devices::GetBlockPerSMCount(){
//	int device;
//	CudaCheck(hipGetDevice(&device));
//	return deviceProp[device].max;
//}

int Devices::GetWarpSize(){
	int device;
	CudaCheck(hipGetDevice(&device));

	return deviceProp[device].warpSize;
}

int Devices::GetBlockHeight(){
	int device;
	CudaCheck(hipGetDevice(&device));

	return _ConvertSMVer2Cores(deviceProp[device].major, deviceProp[device].minor) / GetWarpSize();
}