#include "Utility\CUDA\CUDADevices.cuh"
#include <>
#include <hip/hip_runtime_api.h>
#include "CudaHelper.cuh"

int deviceCount;
hipDeviceProp_t* deviceProp;

void Devices::ExtractDevices(){
	hipError_t error = hipGetDeviceCount(&deviceCount);

	if (deviceCount == 0)
	{
		printf("There are no available device(s) that support CUDA\n");
	}

	deviceProp = new hipDeviceProp_t[deviceCount];

	for (int dev = 0; dev < deviceCount; ++dev){

		CudaCheck(hipSetDevice(dev));
		CudaCheck(hipGetDeviceProperties(&deviceProp[dev], dev));

	}

	CudaCheck(hipSetDevice(0));
}

int Devices::GetCoreCount(){
	int device;
	CudaCheck(hipGetDevice(&device));

	return _ConvertSMVer2Cores(deviceProp[device].major, deviceProp[device].minor) * deviceProp[device].multiProcessorCount;
}

int Devices::GetWarpSize(){
	int device;
	CudaCheck(hipGetDevice(&device));

	return deviceProp[device].warpSize;
}

int Devices::GetBlockHeight(){
	int device;
	CudaCheck(hipGetDevice(&device));

	return _ConvertSMVer2Cores(deviceProp[device].major, deviceProp[device].minor) / GetWarpSize();
}