#include "Utility\CUDAIncludes.h"

#include "Utility\CUDA\CUDADevices.cuh"


int deviceCount;
hipDeviceProp_t* deviceProp;

void Devices::ExtractDevices(){
	hipError_t error = hipGetDeviceCount(&deviceCount);

	if (deviceCount == 0)
	{
		printf("There are no available device(s) that support CUDA\n");
	}

	deviceProp = new hipDeviceProp_t[deviceCount];

	for (int dev = 0; dev < deviceCount; ++dev){

		hipSetDevice(dev);
		hipGetDeviceProperties(&deviceProp[dev], dev);

	}

	hipSetDevice(0);
}

int Devices::GetCoreCount(){
	int device;
	hipGetDevice(&device);

	return _ConvertSMVer2Cores(deviceProp[device].major, deviceProp[device].minor) * deviceProp[device].multiProcessorCount;
}

int Devices::GetWarpSize(){
	int device;
	hipGetDevice(&device);

	return deviceProp[device].warpSize;
}

int Devices::GetBlockHeight(){
	int device;
	hipGetDevice(&device);

	return _ConvertSMVer2Cores(deviceProp[device].major, deviceProp[device].minor) / GetWarpSize();
}