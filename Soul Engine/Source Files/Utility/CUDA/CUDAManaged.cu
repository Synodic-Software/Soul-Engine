#include "Utility\CUDA\CUDAManaged.cuh"


void* Managed::operator new(size_t len){
	void *ptr;
	CudaCheck(hipMallocManaged((void**)&ptr, len));
	hipDeviceSynchronize();
	return ptr;
}

void Managed::operator delete(void *ptr) {
	hipDeviceSynchronize();
	CudaCheck(hipFree(ptr));
}

