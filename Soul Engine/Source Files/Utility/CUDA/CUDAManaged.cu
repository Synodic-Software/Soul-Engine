#include "Utility\CUDA\CUDAManaged.cuh"
#include <hip/hip_runtime.h>

void* Managed::operator new(size_t len){
	void *ptr;
	hipMallocManaged((void**)&ptr, len);
	hipDeviceSynchronize();
	return ptr;
}

void Managed::operator delete(void *ptr) {
	hipDeviceSynchronize();
	hipFree(ptr);
}

