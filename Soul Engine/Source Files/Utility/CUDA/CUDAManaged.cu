#ifndef Managed_H 
#define Managed_H

#include "Engine Core\BasicDependencies.h"

class Managed
{
public:
	void *operator new(size_t len){
		void *ptr;
		hipMallocManaged(&ptr, len);
		hipDeviceSynchronize();
		return ptr;
	}

		void operator delete(void *ptr) {
		hipDeviceSynchronize();
		hipFree(ptr);
	}
};

#endif