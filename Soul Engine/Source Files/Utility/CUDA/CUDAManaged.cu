#include "Utility\CUDA\CUDAManaged.cuh"


void* Managed::operator new(size_t len){
	void *ptr;
	hipMallocManaged(&ptr, len);
	hipDeviceSynchronize();
	return ptr;
}

void Managed::operator delete(void *ptr) {
	hipDeviceSynchronize();
	hipFree(ptr);
}

