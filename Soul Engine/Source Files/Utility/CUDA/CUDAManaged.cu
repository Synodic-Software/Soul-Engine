#pragma once

#include "Engine Core\BasicDependencies.h"
#include "Utility\CUDA\HelperClasses.cuh"

class Managed
{
public:
	void *operator new(size_t len){
		void *ptr;
		hipMallocManaged(&ptr, len);
		hipDeviceSynchronize();
		return ptr;
	}

	void operator delete(void *ptr) {
		hipDeviceSynchronize();
		hipFree(ptr);
	}
};
