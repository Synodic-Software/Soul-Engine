#include "CUDADevices.cuh"
#include <>
#include <hip/hip_runtime_api.h>
#include "Utility/CUDA/CudaHelper.cuh"

int deviceCount;
hipDeviceProp_t* deviceProp;

void ExtractDevices(){
	hipError_t error = hipGetDeviceCount(&deviceCount);

	if (deviceCount == 0)
	{
		return;
	}

	deviceProp = new hipDeviceProp_t[deviceCount];

	for (int dev = 0; dev < deviceCount; ++dev){

		CudaCheck(hipSetDevice(dev));
		CudaCheck(hipGetDeviceProperties(&deviceProp[dev], dev));

	}

}

int GetCoreCount(){
	int device;
	CudaCheck(hipGetDevice(&device));
	return _ConvertSMVer2Cores(deviceProp[device].major, deviceProp[device].minor) * deviceProp[device].multiProcessorCount;
}

int GetSMCount(){
	int device;
	CudaCheck(hipGetDevice(&device));
	return deviceProp[device].multiProcessorCount;
}

int GetWarpSize(){
	int device;
	CudaCheck(hipGetDevice(&device));

	return deviceProp[device].warpSize;
}

int GetBlockHeight(){
	int device;
	CudaCheck(hipGetDevice(&device));

	return _ConvertSMVer2Cores(deviceProp[device].major, deviceProp[device].minor) / GetWarpSize();
}

void Terminate() {
	CudaCheck(hipDeviceReset());
}