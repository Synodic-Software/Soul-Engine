#include "Material.cuh"

Material::Material( std::string texName){


	CudaCheck(hipDeviceSynchronize());
	image.LoadFromFile(texName.c_str(),false,true);
	CudaCheck(hipDeviceSynchronize());

}
Material::~Material(){

}