#include "Material.cuh"
#include "Utility\CUDA\CUDAHelper.cuh"

Material::Material( std::string texName){


	CudaCheck(hipDeviceSynchronize());
	image.LoadFromFile(texName.c_str(),false,true);
	CudaCheck(hipDeviceSynchronize());

}
Material::~Material(){

}