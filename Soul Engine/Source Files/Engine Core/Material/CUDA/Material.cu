#include "hip/hip_runtime.h"
#include "Material.cuh"

Bitmap LoadBmp(const char* filename) {
	Bitmap bmp = Bitmap::bitmapFromFile(filename);
	bmp.flipVertically();
	return bmp;
}


#define MAX(a,b) ((a > b) ? a : b)

uint getMipMapLevels(hipExtent size)
{
	size_t sz = MAX(MAX(size.width, size.height), size.depth);

	uint levels = 0;

	while (sz)
	{
		sz /= 2;
		levels++;
	}

	return levels;
}
inline void _checkHost(bool test, const char *condition, const char *file, int line, const char *func)
{
	if (!test)
	{
		fprintf(stderr, "HOST error at %s:%d (%s) \"%s\" \n",
			file, line, condition, func);
		exit(EXIT_FAILURE);
	}
}

#define checkHost(condition)   _checkHost(condition, #condition,__FILE__,__LINE__,__FUNCTION__)






__device__ __inline__ uchar4 to_uchar4(float4 vec)
{
	return make_uchar4((unsigned char)vec.x, (unsigned char)vec.y, (unsigned char)vec.z, (unsigned char)vec.w);
}




__global__ void
d_mipmap(hipSurfaceObject_t mipOutput, hipTextureObject_t mipInput, uint imageW, uint imageH)
{
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;

	float px = 1.0 / float(imageW);
	float py = 1.0 / float(imageH);


	if ((x < imageW) && (y < imageH))
	{
		// take the average of 4 samples

		// we are using the normalized access to make sure non-power-of-two textures
		// behave well when downsized.
		float4 color =
			(tex2D<float4>(mipInput, (x + 0) * px, (y + 0) * py)) +
			(tex2D<float4>(mipInput, (x + 1) * px, (y + 0) * py)) +
			(tex2D<float4>(mipInput, (x + 1) * px, (y + 1) * py)) +
			(tex2D<float4>(mipInput, (x + 0) * px, (y + 1) * py));


		color.x /= 4.0;
		color.y /= 4.0;
		color.z /= 4.0;
		color.w /= 4.0;
		color.x *= 255.0;
		color.y *= 255.0;
		color.z *= 255.0;
		color.w *= 255.0;
		color = fminf(color, make_float4(255.0, 255.0, 255.0, 255.0));

		surf2Dwrite(to_uchar4(color), mipOutput, x * sizeof(uchar4), y);
	}
}



void generateMipMaps(hipMipmappedArray_t mipmapArray, hipExtent size)
{
	size_t width = size.width;
	size_t height = size.height;


	uint level = 0;

	while (width != 1 || height != 1)
	{
		width /= 2;
		width = MAX((size_t)1, width);
		height /= 2;
		height = MAX((size_t)1, height);

		hipArray_t levelFrom;
		CudaCheck(hipGetMipmappedArrayLevel(&levelFrom, mipmapArray, level));
		hipArray_t levelTo;
		CudaCheck(hipGetMipmappedArrayLevel(&levelTo, mipmapArray, level + 1));

		hipExtent  levelToSize;
		CudaCheck(hipArrayGetInfo(NULL, &levelToSize, NULL, levelTo));
		checkHost(levelToSize.width == width);
		checkHost(levelToSize.height == height);
		checkHost(levelToSize.depth == 0);

		// generate texture object for reading
		hipTextureObject_t         texInput;
		hipResourceDesc            texRes;
		memset(&texRes, 0, sizeof(hipResourceDesc));

		texRes.resType = hipResourceTypeArray;
		texRes.res.array.array = levelFrom;

		hipTextureDesc             texDescr;
		memset(&texDescr, 0, sizeof(hipTextureDesc));

		texDescr.normalizedCoords = 1;
		texDescr.filterMode = hipFilterModeLinear;

		texDescr.addressMode[0] = hipAddressModeClamp;
		texDescr.addressMode[1] = hipAddressModeClamp;
		texDescr.addressMode[2] = hipAddressModeClamp;

		texDescr.readMode = hipReadModeNormalizedFloat;

		CudaCheck(hipCreateTextureObject(&texInput, &texRes, &texDescr, NULL));

		// generate surface object for writing

		hipSurfaceObject_t surfOutput;
		hipResourceDesc    surfRes;
		memset(&surfRes, 0, sizeof(hipResourceDesc));
		surfRes.resType = hipResourceTypeArray;
		surfRes.res.array.array = levelTo;

		CudaCheck(hipCreateSurfaceObject(&surfOutput, &surfRes));

		// run mipmap kernel
		dim3 blockSize(16, 16, 1);
		dim3 gridSize(((uint)width + blockSize.x - 1) / blockSize.x, ((uint)height + blockSize.y - 1) / blockSize.y, 1);

		d_mipmap << <gridSize, blockSize >> >(surfOutput, texInput, (uint)width, (uint)height);

		CudaCheck(hipDeviceSynchronize());
		CudaCheck(hipGetLastError());

		CudaCheck(hipDestroySurfaceObject(surfOutput));

		CudaCheck(hipDestroyTextureObject(texInput));

		level++;
	}
}


Material::Material( std::string texName){

	//hipTextureObject_t texObj;
	//hipArray* cuArray;
	//Bitmap bmp;

	//CudaCheck(hipDeviceSynchronize());
	//bmp = LoadBmp(texName.c_str());


	//Image image;
	//image.size = make_hipExtent(bmp.width(), bmp.height(), 0);
	//image.h_data = bmp.pixelBuffer();


	//uint levels = getMipMapLevels(image.size);

	//////Allocate CUDA array in device memory 
	//std::cout << bmp.height() << " " << bmp.width() << " " << sizeof(float) << std::endl;

	////for (int i = 0; i < bmp.width(); i++){
	////	for (int c = 0; c < bmp.height(); c++){
	////		std::cout << (short)bmp.pixelBuffer()[c] << " ";
	////	}
	////	std::cout << std::endl;
	////}


	//hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
	//CudaCheck(hipMallocMipmappedArray(&image.mipmapArray, &desc, image.size, levels));

	//hipArray_t level0;
	//CudaCheck(hipGetMipmappedArrayLevel(&level0, image.mipmapArray, 0));

	//hipMemcpy3DParms copyParams = { 0 };
	//copyParams.srcPtr = make_hipPitchedPtr(image.h_data, image.size.width * sizeof(uchar4), image.size.width, image.size.height);
	//copyParams.dstArray = level0;
	//copyParams.extent = image.size;
	//copyParams.extent.depth = 1;
	//copyParams.kind = hipMemcpyHostToDevice;
	//CudaCheck(hipMemcpy3D(&copyParams));

	//// compute rest of mipmaps based on level 0
	//generateMipMaps(image.mipmapArray, image.size);

	//// generate bindless texture object

	//hipResourceDesc            resDescr;
	//memset(&resDescr, 0, sizeof(hipResourceDesc));

	//resDescr.resType = hipResourceTypeMipmappedArray;
	//resDescr.res.mipmap.mipmap = image.mipmapArray;

	//hipTextureDesc             texDescr;
	//memset(&texDescr, 0, sizeof(hipTextureDesc));

	//texDescr.normalizedCoords = 1;
	//texDescr.filterMode = hipFilterModeLinear;
	//texDescr.mipmapFilterMode = hipFilterModeLinear;

	//texDescr.addressMode[0] = hipAddressModeWrap;
	//texDescr.addressMode[1] = hipAddressModeWrap;
	//texDescr.addressMode[2] = hipAddressModeWrap;

	//texDescr.maxMipmapLevelClamp = float(levels - 1);

	//texDescr.readMode = hipReadModeNormalizedFloat;

	//CudaCheck(hipCreateTextureObject(&texObj, &resDescr, &texDescr, NULL));

	//CudaCheck(hipDeviceSynchronize());
}
Material::~Material(){

	// Destroy texture object 
	//hipDestroyTextureObject(texObj); 

	//// Free device memory 
	//hipFree(cuArray);

	//make a cleanup function

	//if (texture != NULL){
	//	delete texture;
	//}
	//if (defaultTexture!=NULL){
	//	delete defaultTexture;
	//}
}
//void Material::SetTexture(std::string name ){
//	//Bitmap bmp = Bitmap::bitmapFromFile(name);
//		//bmp.flipVertically();
//		//texture=new Texture(bmp);
//		//textureHandle = glGetTextureHandleARB(texture->object());
//	//textureIsLoaded = true;
//}
//void Material::SetDefaultTexture(std::string name){
//	//Bitmap bmp = Bitmap::bitmapFromFile(name);
//	//bmp.flipVertically(); 
//	//defaultTexture = new Texture(bmp);
//}