#include "hip/hip_runtime.h"
#include "Engine Core/Camera/CUDA/Camera.cuh"
#include <glm/gtx/rotate_vector.hpp>

__host__ __device__ Camera::Camera() :
	resolution(0,0),
	aspectRatio(0),
    position(0.0f,0.0f,0.0f),
	forward(0.0f,0.0f,1.0f),
	right(1.0f, 0.0f, 0.0f),
	fieldOfView(90.0f,65.0f),
	aperture(2*MILLIMETER),
	focalDistance(17*MILLIMETER),
	circularDistribution(false)
{	
}

__host__ __device__  Camera::~Camera(){

}

__host__ __device__ void Camera::SetAspect(float newA){
	aspectRatio = newA;
}
__host__ __device__ float Camera::GetAspect(){
	return aspectRatio;
}

__host__ __device__ glm::vec3 Camera::Position() const {
    return position;
}

__host__ __device__ void Camera::SetPosition(const glm::vec3& positionN) {
    position = positionN;
}

__host__ __device__ void Camera::OffsetPosition(const glm::vec3& offset) {
    position += offset;
}

__host__ __device__ glm::vec2 Camera::FieldOfView() const{
    return fieldOfView;
}
__host__ __device__ void Camera::SetFieldOfView(glm::vec2 fieldOfView) {
    fieldOfView = fieldOfView;
}



__host__ __device__ glm::vec3 Camera::Forward() const {
    return forward;
}
__host__ __device__ void Camera::SetForward(glm::vec3& forN){
	forward = glm::normalize(forN);
}

__host__ __device__ glm::vec3 Camera::Right() const {
    return right;
}
__host__ __device__ void Camera::SetRight(glm::vec3& rightn) {
	right = normalize(rightn);
}


__device__ void Camera::SetupRay(uint& index, Ray& ray, hiprandState& rng){

	//OPTIMIZED! int x = index - (y*resolution.y);

	// generate random jitter offsets for supersampled antialiasing

	//OPTIMIZED! float jitterValueX = uniformDistribution(rng) - 0.5f;
	//OPTIMIZED! float jitterValueY = uniformDistribution(rng) - 0.5f;

	// compute important values

	// compute point on image plane

	//OPTIMIZED! glm::vec3 middle = position + forward;
	//OPTIMIZED! glm::vec3 horizontal = right * tan(glm::radians(fieldOfView.x * 0.5f));
	//OPTIMIZED! glm::vec3 vertical = verticalAxis * tan((glm::radians(-fieldOfView.y * 0.5f))); 

	// move and resize image plane based on focalDistance
	// could also incorporate this into the original computations of the point

	//OPTIMIZED! glm::vec3 pointOnImagePlane = position + ((((position + forward) + (((2 * sx) - 1) *
	//	(right * tan(glm::radians(fieldOfView.x * 0.5f)))) + (((2 * sy) - 1) *
	//	(verticalAxis * tan((glm::radians(-fieldOfView.y * 0.5f)))))) - position) * focalDistance); // Important for depth of field!

	// now compute the point on the aperture (or lens)

	// generate random numbers for sampling a point on the aperture

	//OPTIMIZED! float random1 = uniformDistribution(rng);
	//OPTIMIZED! float random2 = uniformDistribution(rng);

	// sample a point on the circular aperture

	//OPTIMIZED! float apertureX = cos(angle) * distance;
	//OPTIMIZED! float apertureY = sin(angle) * distance;

	

	//OPTIMIZED!glm::vec3 apertureToImagePlane = (position + ((((position + forward) + (((2 * sx) - 1) *
	//	(right * tan(glm::radians(fieldOfView.x * 0.5f)))) + (((2 * sy) - 1) *
	//	(verticalAxis * tan((glm::radians(-fieldOfView.y * 0.5f)))))) - position) * focalDistance)) - aperturePoint;




	uint y = index / resolution.x;

	float sx = ((hiprand_uniform(&rng) - 0.5f) + (index %resolution.x)) / (resolution.x - 1);
	float sy = ((hiprand_uniform(&rng) - 0.5f) + y) / (resolution.y - 1);

	float angle = TWO_PI * hiprand_uniform(&rng);
	float distance = aperture * sqrt(hiprand_uniform(&rng));


	//ALTERNATE aperaturPoint
	//+ ((cos(angle) * distance) * right) + ((sin(angle) * distance) * verticalAxis)

	glm::vec3 aperturePoint = position ;
	
	glm::vec3 pointOnPlaneOneUnitAwayFromEye = 
		(aperturePoint + forward) + (((2 * sx) - 1) * xHelper) + (((2 * sy) - 1) * yHelper);

	ray.origin = glm::vec4(aperturePoint.x, aperturePoint.y, aperturePoint.z, 0.0f);
	glm::vec3 tmp= glm::normalize((position + ((pointOnPlaneOneUnitAwayFromEye-position) * focalDistance)) - aperturePoint);
	ray.direction = glm::vec4(tmp.x, tmp.y, tmp.z, 40000000000000000000.0f);

}

__host__ __device__ void Camera::UpdateVariables(){
	verticalAxis = normalize(cross(right, forward));
	
	yHelper=verticalAxis * tan((glm::radians(-fieldOfView.y * 0.5f)));
	xHelper= right * tan(glm::radians(fieldOfView.x * 0.5f));
}

__host__ __device__ bool Camera::IsViewable() const{
	return !circularDistribution;
}
__host__ __device__ void Camera::SetCircle(bool cir){
	circularDistribution = cir;
}
__host__ __device__ void Camera::OffsetOrientation(float x, float y){
	right = normalize(glm::rotateY(right, glm::radians(x)));
	forward = normalize(glm::rotateY(forward, glm::radians(x)));

	forward = normalize(glm::rotate(forward, glm::radians(y),right));
}